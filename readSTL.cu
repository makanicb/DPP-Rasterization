#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <limits>
#include <cmath>
/*
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/tuple.h>
#include <thrust/copy.h>
*/

#include<viskores/cont/ArrayHandle.h>
#include<viskores/Types.h>

#include "readSTL.h"

/*
    readBinarySTL: A function for reading a binary STL file to standard output

    Parameters:
    * filename: a pointer to an array of characters containing the
    	name of the file to read

    Returns 0 on sucess
*/
int readBinarySTL (char *filename)
{
	//open the stl file in read mode
	FILE *f = fopen(filename, "r");
	//create buffers to store the header of the file and the
	//number of triangles
	char header[80];
	unsigned int numTri = 0;
	//read from the file into the buffers
	fread(header, 1, 80, f);
	fread(&numTri, 4, 1, f);
	//print the values in the buffers
	std::cout << header << std::endl;
	std::cout << numTri << std::endl;
	//read the triangles
	//create buffers for the normal vector, position vectors
	//for the three vertices of each triangle, and attribute
	//byte count
	float normal[3];
	float vert1[3];
	float vert2[3];
	float vert3[3];
	short attr;
	for (unsigned int i = 0; i < numTri; i++)
	{
		//read into the buffers
		fread(normal, 4, 3, f);
		fread(vert1, 4, 3, f);
		fread(vert2, 4, 3, f);
		fread(vert3, 4, 3, f);
		fread(&attr, 2, 1, f);
		//print out the values in the buffers
		std::cout << "N\tV1\tV2\tV3" << std::endl;
		for(int j = 0; j < 3; j++)
		{
			std::cout << normal[j] << "\t";
			std::cout << vert1[j] << "\t";
			std::cout << vert2[j] << "\t";
			std::cout << vert3[j] << "\t";
			std::cout << std::endl;
		}
		std::cout << "Attribute Byte Count: " << attr << std::endl;
	}
	//close the file
	fclose(f);
	return 0;
}

/*
    getNumTriSTL: Get the number of triangles encoded within an STL file

    Parameters:
    * filename: a pointer to an array of characters containing the name
    	of the STL file to read

    Returns the number of triangles in the file
*/
unsigned int getNumTriSTL(char *filename)
{
	//open the STL file
	FILE *f = fopen(filename, "r");
	//go to the end of the STL file header
	fseek(f, 80, SEEK_SET);
	//create a buffer to read the number of triangles into
	unsigned int numTri = 0;
	//read the number of triangles into the buffer
	fread(&numTri, 4, 1, f);
	//close the STL file
	fclose(f);
	//return the number of triangles
	return numTri;
}

template<typename WritePortalType>
int getColor(float* norm, WritePortalType &color, int index)
{
	float light[3] = {0, 0, 1};
	float mag = std::sqrt(norm[0] * norm[0] + norm[1] * norm[1] + norm[2] * norm[2]);
	float dot = (norm[0] * light[0] + norm[1] * light[1] + norm[2] * light[2]) / mag;
	dot = std::max(dot, 0.0f);
	color.Set(index, viskores::make_Vec((char)(255 * dot), (char)(255 * dot), (char)(255 * dot)));
	/*if(dot > 1)
	{
		std::cout << std::endl;
		std::cout << norm[0] << ", " << norm[1] << ", " << norm[2] << std::endl;
		std::cout << dot << ", " << mag << std::endl;
	}*/
	//std::cout << 255 * dot << ", " << (int) (255 * dot) << ", " << (int)(unsigned char)(char) (255 * dot) << std::endl;
	return 0;
}

/*
    readTriFromBinarySTL: read the triangles of an STL file into buffers

    Parameters:
    * p1: the address of the buffer to store the positions of the first
    	point of each triangle in
    * p2: the address of the buffer to store the positions of the
    	second point of each triangle in
    * p3: the address of the buffer to store the positions of the third
    	point of each triangle in
    * color: the address of the buffer to store the color of each
    	triangle in
    * filename: the name of the file to read

    Returns the number of triangles read
*/

unsigned int readTriFromBinarySTL(
	viskores::cont::ArrayHandle<viskores::Vec3f> &p1,
	viskores::cont::ArrayHandle<viskores::Vec3f> &p2,
	viskores::cont::ArrayHandle<viskores::Vec3f> &p3,
	viskores::cont::ArrayHandle<viskores::Vec3ui_8> &color,
	char *filename, int &width, int &height, int scale)
{
	//get the number of triangles to read
	unsigned int numTri = getNumTriSTL(filename);
	unsigned int subdivisions = 1;
	unsigned int subdividedNumTri = numTri * pow(4, subdivisions); 
	//resize arrays
	p1.Allocate(subdividedNumTri);
	p2.Allocate(subdividedNumTri);
	p3.Allocate(subdividedNumTri);
	color.Allocate(subdividedNumTri);
	//create writers
	auto p1_Writer = p1.WritePortal();
	auto p2_Writer = p2.WritePortal();
	auto p3_Writer = p3.WritePortal();
	auto color_Writer = color.WritePortal();
	//open the file
	FILE *f = fopen(filename, "r");
	//go to the start of the triangle information
	fseek(f, 84, SEEK_SET);
	//iterate over the triangles
	unsigned int i = 0;
	//create buffers for each bit of triangle information
	float norm[3];
	float v1[3];
	float v2[3];
	float v3[3];
	short attr;
	//iterate over triangles for max and min values
	//variables for tracking lowest values
	int lowx = std::numeric_limits<int>::max();
	int lowy = std::numeric_limits<int>::max();
	//reset width and height to minimum possible value
	width = std::numeric_limits<int>::min();
	height = std::numeric_limits<int>::min();
	for(; i < numTri; i++)
	{
		//read into buffers
		fread(norm, 4, 3, f);
		fread(v1, 4, 3, f);
		fread(v2, 4, 3, f);
		fread(v3, 4, 3, f);
		fread(&attr, 2, 1, f);
		//scale vertices
		for(int j = 0; j < 3; j++)
		{
			v1[j] *= scale;
			v2[j] *= scale;
			v3[j] *= scale;
		}
		//process buffers
		width = std::max(width, std::max((int) v1[0], std::max((int) v2[0], (int) v3[0])) + 1);
		height = std::max(height, std::max((int) v1[1], std::max((int) v2[1], (int) v3[1])) + 1);
		lowx = std::min(lowx, std::min((int) v1[0], std::min((int) v2[0], (int) v3[0])));
		lowy = std::min(lowy, std::min((int) v1[1], std::min((int) v2[1], (int) v3[1])));
	}
	// Move corner to origin
	width -= lowx;
	height -= lowy;
	//std::cout << "W: " << width << " H: " << height << std::endl;
	//std::cout << "LOWX: " << lowx << " LOWY: " << lowy << std::endl;

	fseek(f, 84, SEEK_SET); //go back to the start of the file

	//iterate over the triangles to read into triangle buffers
	
	//std::cout << "x, y, z" << std::endl;
	for(i = 0; i < numTri; i++)
	{
		//read into buffers
		fread(norm, 4, 3, f);
		fread(v1, 4, 3, f);
		fread(v2, 4, 3, f);
		fread(v3, 4, 3, f);
		fread(&attr, 2, 1, f);
		//scale vertices
		for(int j = 0; j < 3; j++)
		{
			v1[j] *= scale;
			v2[j] *= scale;
			v3[j] *= scale;
		}
		//process buffers
		//std::cout << v1[0] - lowx << ", " << v1[1] - lowy << ", " << v1[2] << std::endl;
		p1_Writer.Set(i, viskores::make_Vec(v1[0] - lowx, v1[1] - lowy, v1[2]));
		//std::cout << v2[0] - lowx << ", " << v2[1] - lowy << ", " << v2[2] << std::endl;
		p2_Writer.Set(i, viskores::make_Vec(v2[0] - lowx, v2[1] - lowy, v2[2]));
		//std::cout << v3[0] - lowx << ", " << v3[1] - lowy << ", " << v3[2] << std::endl;
		p3_Writer.Set(i, viskores::make_Vec(v3[0] - lowx, v3[1] - lowy, v3[2]));
		getColor(norm, color_Writer, i);
	}
	//std::cout << std::endl;

	// Subdivide
	unsigned int curNumTri = i;
	for (int j = 0; j < 1; j++)
	{
		// Create temporary arrays to subdivide into
		viskores::cont::ArrayHandle<viskores::Vec3f> tmp_p1;
		viskores::cont::ArrayHandle<viskores::Vec3f> tmp_p2;
		viskores::cont::ArrayHandle<viskores::Vec3f> tmp_p3;
		viskores::cont::ArrayHandle<viskores::Vec3ui_8> tmp_color;

		// Copy point arrays
		tmp_p1.Allocate(subdividedNumTri);
		tmp_p2.Allocate(subdividedNumTri);
		tmp_p3.Allocate(subdividedNumTri);
		tmp_color.Allocate(subdividedNumTri);

		// Create readers
		auto p1_Reader = p1.ReadPortal();
		auto p2_Reader = p2.ReadPortal();
		auto p3_Reader = p3.ReadPortal();
		auto color_Reader = color.ReadPortal();

		// Create writers	
		auto tmp1_Writer = tmp_p1.WritePortal();
		auto tmp2_Writer = tmp_p2.WritePortal();
		auto tmp3_Writer = tmp_p3.WritePortal();
		auto tmpC_Writer = tmp_color.WritePortal();

		// Generate subdivisions
		for (unsigned int k = 0; k < curNumTri; k++)
		{
			// Get original positions
			viskores::Vec3f v1 = p1_Reader.Get(k);
			viskores::Vec3f v2 = p2_Reader.Get(k);
			viskores::Vec3f v3 = p3_Reader.Get(k);
			viskores::Vec3ui_8 col = color_Reader.Get(k);

			// Calculate midpoints
			viskores::Vec3f m12 = (v1 + v2) * 0.5f;
			viskores::Vec3f m13 = (v1 + v3) * 0.5f;
			viskores::Vec3f m23 = (v2 + v3) * 0.5f;

			// First triangle
			tmp1_Writer.Set(k, v1);
			tmp2_Writer.Set(k, m12);
			tmp3_Writer.Set(k, m13);
			tmpC_Writer.Set(k, col);

			// Second triangle
			tmp1_Writer.Set(k+1, m12);
			tmp2_Writer.Set(k+1, v2);
			tmp3_Writer.Set(k+1, m23);
			tmpC_Writer.Set(k+1, col);

			// Third triangle
			tmp1_Writer.Set(k+2, m13);
			tmp2_Writer.Set(k+2, m23);
			tmp3_Writer.Set(k+2, v3);
			tmpC_Writer.Set(k+2, col);

			// Fourth triangle
			tmp1_Writer.Set(k+3, m13);
			tmp2_Writer.Set(k+3, m12);
			tmp3_Writer.Set(k+3, m23);
			tmpC_Writer.Set(k+3, col);

		}

		//Copy subdivisions to original vectors
		p1.DeepCopyFrom(tmp_p1);
		p2.DeepCopyFrom(tmp_p2);
		p3.DeepCopyFrom(tmp_p3);
		color.DeepCopyFrom(tmp_color);

		curNumTri *= 4;
	}

	//copy host vectors into device vectors
	//thrust::copy(p1.begin(), p1.end(), hp1.begin());
	return curNumTri;
}

/*int main(int argc, char **argv)
{
	readBinarySTL(argv[1]);
	unsigned int numTri = getNumTriSTL(argv[1]);
	std::cout << numTri << std::endl;
	thrust::device_vector<thrust::tuple<float,float,float>> p1(numTri);
	thrust::device_vector<thrust::tuple<float,float,float>> p2(numTri);
	thrust::device_vector<thrust::tuple<float,float,float>> p3(numTri);
	thrust::device_vector<thrust::tuple<char,char,char>> color(numTri);
	readTriFromBinarySTL(p1, p2, p3, color, argv[1]);
	thrust::host_vector<thrust::tuple<float,float,float>> temp = p1;
	for(int i = 0; i < temp.size(); i++)
		std::cout << temp[i].get<0>() << "\t";
	std::cout << std::endl;
	return 0;
}*/
