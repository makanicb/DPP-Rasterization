
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <iostream>
/*
    A function for reading a binary STL file to standard output
    Parameters:
    - filename: a pointer to an array of characters containing the
      name of the file to read
*/
int readBinarySTL (char *filename)
{
	//open the stl file in read mode
	FILE *f = fopen(filename, "r");
	//create buffers to store the header of the file and the
	//number of triangles
	char header[80];
	unsigned int numTri = 0;
	//read from the file into the buffers
	fread(header, 1, 80, f);
	fread(&numTri, 4, 1, f);
	//print the values in the buffers
	std::cout << header << std::endl;
	std::cout << numTri << std::endl;
	//close the file
	fclose(f);
	return 0;
}

int main(int argc, char **argv)
{
	readBinarySTL(argv[1]);
	return 0;
}
