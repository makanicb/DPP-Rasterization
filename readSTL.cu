
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <iostream>
/*
    A function for reading a binary STL file to standard output
    Parameters:
    - filename: a pointer to an array of characters containing the
      name of the file to read
*/
int readBinarySTL (char *filename)
{
	//open the stl file in read mode
	FILE *f = fopen(filename, "r");
	//create buffers to store the header of the file and the
	//number of triangles
	char header[80];
	unsigned int numTri = 0;
	//read from the file into the buffers
	fread(header, 1, 80, f);
	fread(&numTri, 4, 1, f);
	//print the values in the buffers
	std::cout << header << std::endl;
	std::cout << numTri << std::endl;
	//read the triangles
	//create buffers for the normal vector, position vectors
	//for the three vertices of each triangle, and attribute
	//byte count
	float normal[3];
	float vert1[3];
	float vert2[3];
	float vert3[3];
	short attr;
	for (int i = 0; i < numTri; i++)
	{
		//read into the buffers
		fread(normal, 4, 3, f);
		fread(vert1, 4, 3, f);
		fread(vert2, 4, 3, f);
		fread(vert3, 4, 3, f);
		fread(&attr, 2, 1, f);
		//print out the values in the buffers
		std::cout << "N\tV1\tV2\tV3" << std::endl;
		for(int j = 0; j < 3; j++)
		{
			std::cout << normal[j] << "\t";
			std::cout << vert1[j] << "\t";
			std::cout << vert2[j] << "\t";
			std::cout << vert3[j] << "\t";
			std::cout << std::endl;
		}
		std::cout << "Attribute Byte Count: " << attr << std::endl;
	}
	//close the file
	fclose(f);
	return 0;
}

int main(int argc, char **argv)
{
	readBinarySTL(argv[1]);
	return 0;
}
