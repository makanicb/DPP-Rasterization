#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/tuple.h>
#include <thrust/copy.h>

#include "readSTL.h"

/*
    readBinarySTL: A function for reading a binary STL file to standard output

    Parameters:
    * filename: a pointer to an array of characters containing the
    	name of the file to read

    Returns 0 on sucess
*/
int readBinarySTL (char *filename)
{
	//open the stl file in read mode
	FILE *f = fopen(filename, "r");
	//create buffers to store the header of the file and the
	//number of triangles
	char header[80];
	unsigned int numTri = 0;
	//read from the file into the buffers
	fread(header, 1, 80, f);
	fread(&numTri, 4, 1, f);
	//print the values in the buffers
	std::cout << header << std::endl;
	std::cout << numTri << std::endl;
	//read the triangles
	//create buffers for the normal vector, position vectors
	//for the three vertices of each triangle, and attribute
	//byte count
	float normal[3];
	float vert1[3];
	float vert2[3];
	float vert3[3];
	short attr;
	for (int i = 0; i < numTri; i++)
	{
		//read into the buffers
		fread(normal, 4, 3, f);
		fread(vert1, 4, 3, f);
		fread(vert2, 4, 3, f);
		fread(vert3, 4, 3, f);
		fread(&attr, 2, 1, f);
		//print out the values in the buffers
		std::cout << "N\tV1\tV2\tV3" << std::endl;
		for(int j = 0; j < 3; j++)
		{
			std::cout << normal[j] << "\t";
			std::cout << vert1[j] << "\t";
			std::cout << vert2[j] << "\t";
			std::cout << vert3[j] << "\t";
			std::cout << std::endl;
		}
		std::cout << "Attribute Byte Count: " << attr << std::endl;
	}
	//close the file
	fclose(f);
	return 0;
}

/*
    getNumTriSTL: Get the number of triangles encoded within an STL file

    Parameters:
    * filename: a pointer to an array of characters containing the name
    	of the STL file to read

    Returns the number of triangles in the file
*/
unsigned int getNumTriSTL(char *filename)
{
	//open the STL file
	FILE *f = fopen(filename, "r");
	//go to the end of the STL file header
	fseek(f, 80, SEEK_SET);
	std::cout << ftell(f) << std::endl;
	//create a buffer to read the number of triangles into
	unsigned int numTri = 0;
	//read the number of triangles into the buffer
	fread(&numTri, 4, 1, f);
	//close the STL file
	fclose(f);
	//return the number of triangles
	return numTri;
}

thrust::tuple<char,char,char> getColor(float* norm)
{
	float light[3] = {0, 0, 1};
	float dot = norm[0] * light[0] + norm[1] * light[1] + norm[2] * light[2];
	dot = max(dot, 0.0);
	return thrust::make_tuple((char)(255 * dot), (char)(255 * dot), (char)(255 * dot));
}

/*
    readTriFromBinarySTL: read the triangles of an STL file into buffers

    Parameters:
    * p1: the address of the buffer to store the positions of the first
    	point of each triangle in
    * p2: the address of the buffer to store the positions of the
    	second point of each triangle in
    * p3: the address of the buffer to store the positions of the third
    	point of each triangle in
    * color: the address of the buffer to store the color of each
    	triangle in
    * filename: the name of the file to read

    Returns the number of triangles read
*/

unsigned int readTriFromBinarySTL(
	thrust::device_vector<thrust::tuple<float,float,float>> &p1,
	thrust::device_vector<thrust::tuple<float,float,float>> &p2,
	thrust::device_vector<thrust::tuple<float,float,float>> &p3,
	thrust::device_vector<thrust::tuple<char,char,char>> &color,
	char *filename)
{
	//get the number of triangles to read
	unsigned int numTri = getNumTriSTL(filename);
	//create host buffers to store data temporarily
	thrust::host_vector<thrust::tuple<float,float,float>> hp1(numTri);
	thrust::host_vector<thrust::tuple<float,float,float>> hp2(numTri);
	thrust::host_vector<thrust::tuple<float,float,float>> hp3(numTri);
	thrust::host_vector<thrust::tuple<char,char,char>> hcolor(numTri);
	//open the file
	FILE *f = fopen(filename, "r");
	//go to the start of the triangle information
	fseek(f, 84, SEEK_SET);
	//iterate over the triangles
	unsigned int i = 0;
	//create buffers for each bit of triangle information
	float norm[3];
	float v1[3];
	float v2[3];
	float v3[3];
	short attr;
	//iterate
	for(; i < numTri; i++)
	{
		//read into buffers
		fread(norm, 4, 3, f);
		fread(v1, 4, 3, f);
		fread(v2, 4, 3, f);
		fread(v3, 4, 3, f);
		fread(&attr, 2, 1, f);
		//process buffers
		hp1[i] = thrust::make_tuple(v1[0], v1[1], v1[2]);
		hp2[i] = thrust::make_tuple(v2[0], v2[1], v2[2]);
		hp3[i] = thrust::make_tuple(v3[0], v3[1], v3[2]);
		hcolor[i] = getColor(norm);
	}

	//copy host vectors into device vectors
	//thrust::copy(p1.begin(), p1.end(), hp1.begin());
	p1 = hp1;
	p2 = hp2;
	p3 = hp3;
	color = hcolor;

	return i;
}

int main(int argc, char **argv)
{
	readBinarySTL(argv[1]);
	unsigned int numTri = getNumTriSTL(argv[1]);
	std::cout << numTri << std::endl;
	thrust::device_vector<thrust::tuple<float,float,float>> p1(numTri);
	thrust::device_vector<thrust::tuple<float,float,float>> p2(numTri);
	thrust::device_vector<thrust::tuple<float,float,float>> p3(numTri);
	thrust::device_vector<thrust::tuple<char,char,char>> color(numTri);
	readTriFromBinarySTL(p1, p2, p3, color, argv[1]);
	thrust::host_vector<thrust::tuple<float,float,float>> temp = p1;
	for(int i = 0; i < temp.size(); i++)
		std::cout << temp[i].get<0>() << "\t";
	std::cout << std::endl;
	return 0;
}
