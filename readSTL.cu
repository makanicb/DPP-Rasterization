#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <limits>
/*
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/tuple.h>
#include <thrust/copy.h>
*/

#include<viskores/cont/ArrayHandle.h>
#include<viskores/Types.h>

#include "readSTL.h"

/*
    readBinarySTL: A function for reading a binary STL file to standard output

    Parameters:
    * filename: a pointer to an array of characters containing the
    	name of the file to read

    Returns 0 on sucess
*/
int readBinarySTL (char *filename)
{
	//open the stl file in read mode
	FILE *f = fopen(filename, "r");
	//create buffers to store the header of the file and the
	//number of triangles
	char header[80];
	unsigned int numTri = 0;
	//read from the file into the buffers
	fread(header, 1, 80, f);
	fread(&numTri, 4, 1, f);
	//print the values in the buffers
	std::cout << header << std::endl;
	std::cout << numTri << std::endl;
	//read the triangles
	//create buffers for the normal vector, position vectors
	//for the three vertices of each triangle, and attribute
	//byte count
	float normal[3];
	float vert1[3];
	float vert2[3];
	float vert3[3];
	short attr;
	for (unsigned int i = 0; i < numTri; i++)
	{
		//read into the buffers
		fread(normal, 4, 3, f);
		fread(vert1, 4, 3, f);
		fread(vert2, 4, 3, f);
		fread(vert3, 4, 3, f);
		fread(&attr, 2, 1, f);
		//print out the values in the buffers
		std::cout << "N\tV1\tV2\tV3" << std::endl;
		for(int j = 0; j < 3; j++)
		{
			std::cout << normal[j] << "\t";
			std::cout << vert1[j] << "\t";
			std::cout << vert2[j] << "\t";
			std::cout << vert3[j] << "\t";
			std::cout << std::endl;
		}
		std::cout << "Attribute Byte Count: " << attr << std::endl;
	}
	//close the file
	fclose(f);
	return 0;
}

/*
    getNumTriSTL: Get the number of triangles encoded within an STL file

    Parameters:
    * filename: a pointer to an array of characters containing the name
    	of the STL file to read

    Returns the number of triangles in the file
*/
unsigned int getNumTriSTL(char *filename)
{
	//open the STL file
	FILE *f = fopen(filename, "r");
	//go to the end of the STL file header
	fseek(f, 80, SEEK_SET);
	//create a buffer to read the number of triangles into
	unsigned int numTri = 0;
	//read the number of triangles into the buffer
	fread(&numTri, 4, 1, f);
	//close the STL file
	fclose(f);
	//return the number of triangles
	return numTri;
}

template<typename WritePortalType>
int getColor(float* norm, WritePortalType &color, int index)
{
	float light[3] = {0, 0, 1};
	float mag = std::sqrt(norm[0] * norm[0] + norm[1] * norm[1] + norm[2] * norm[2]);
	float dot = (norm[0] * light[0] + norm[1] * light[1] + norm[2] * light[2]) / mag;
	dot = std::max(dot, 0.0f);
	color.Set(index, viskores::make_Vec((char)(255 * dot), (char)(255 * dot), (char)(255 * dot)));
	/*if(dot > 1)
	{
		std::cout << std::endl;
		std::cout << norm[0] << ", " << norm[1] << ", " << norm[2] << std::endl;
		std::cout << dot << ", " << mag << std::endl;
	}*/
	//std::cout << 255 * dot << ", " << (int) (255 * dot) << ", " << (int)(unsigned char)(char) (255 * dot) << std::endl;
	return 0;
}

/*
    readTriFromBinarySTL: read the triangles of an STL file into buffers

    Parameters:
    * p1: the address of the buffer to store the positions of the first
    	point of each triangle in
    * p2: the address of the buffer to store the positions of the
    	second point of each triangle in
    * p3: the address of the buffer to store the positions of the third
    	point of each triangle in
    * color: the address of the buffer to store the color of each
    	triangle in
    * filename: the name of the file to read

    Returns the number of triangles read
*/

unsigned int readTriFromBinarySTL(
	viskores::cont::ArrayHandle<viskores::Vec3f> &p1,
	viskores::cont::ArrayHandle<viskores::Vec3f> &p2,
	viskores::cont::ArrayHandle<viskores::Vec3f> &p3,
	viskores::cont::ArrayHandle<viskores::Vec3ui_8> &color,
	char *filename, int &width, int &height, int scale)
{
	//get the number of triangles to read
	unsigned int numTri = getNumTriSTL(filename);
	//resize arrays
	p1.Allocate(numTri);
	p2.Allocate(numTri);
	p3.Allocate(numTri);
	color.Allocate(numTri);
	//create writers
	auto p1_Writer = p1.ReadWritePortal();
	auto p2_Writer = p2.ReadWritePortal();
	auto p3_Writer = p3.ReadWritePortal();
	auto color_Writer = color.WritePortal();
	//open the file
	FILE *f = fopen(filename, "r");
	//go to the start of the triangle information
	fseek(f, 84, SEEK_SET);
	//iterate over the triangles
	unsigned int i = 0;
	//create buffers for each bit of triangle information
	float norm[3];
	float v1[3];
	float v2[3];
	float v3[3];
	short attr;
	//iterate over triangles for max and min values
	//variables for tracking lowest values
	int lowx = std::numeric_limits<int>::max();
	int lowy = std::numeric_limits<int>::max();
	//reset width and height to minimum possible value
	width = std::numeric_limits<int>::min();
	height = std::numeric_limits<int>::min();
	for(; i < numTri; i++)
	{
		//read into buffers
		fread(norm, 4, 3, f);
		fread(v1, 4, 3, f);
		fread(v2, 4, 3, f);
		fread(v3, 4, 3, f);
		fread(&attr, 2, 1, f);
		//process buffers
		width = std::max(width, std::max((int) v1[0], std::max((int) v2[0], (int) v3[0])) + 1);
		height = std::max(height, std::max((int) v1[1], std::max((int) v2[1], (int) v3[1])) + 1);
		lowx = std::min(lowx, std::min((int) v1[0], std::min((int) v2[0], (int) v3[0])));
		lowy = std::min(lowy, std::min((int) v1[1], std::min((int) v2[1], (int) v3[1])));
	}
	// Move corner to origin
	width -= lowx;
	height -= lowy;
	// Scale size
	width *= scale;
	height *= scale;
	//std::cout << "LOWX: " << lowx << " LOWY: " << lowy << std::endl;

	fseek(f, 84, SEEK_SET); //go back to the start of the file

	//iterate over the triangles to read into triangle buffers
	
	//std::cout << "x, y, z" << std::endl;
	for(i = 0; i < numTri; i++)
	{
		//read into buffers
		fread(norm, 4, 3, f);
		fread(v1, 4, 3, f);
		fread(v2, 4, 3, f);
		fread(v3, 4, 3, f);
		fread(&attr, 2, 1, f);
		//process buffers
		//std::cout << v1[0] - lowx << ", " << v1[1] - lowy << ", " << v1[2] << std::endl;
		p1_Writer.Set(i, viskores::make_Vec(v1[0] - lowx, v1[1] - lowy, v1[2]));
		//std::cout << v2[0] - lowx << ", " << v2[1] - lowy << ", " << v2[2] << std::endl;
		p2_Writer.Set(i, viskores::make_Vec(v2[0] - lowx, v2[1] - lowy, v2[2]));
		//std::cout << v3[0] - lowx << ", " << v3[1] - lowy << ", " << v3[2] << std::endl;
		p3_Writer.Set(i, viskores::make_Vec(v3[0] - lowx, v3[1] - lowy, v3[2]));
		getColor(norm, color_Writer, i);
	}
	//std::cout << std::endl;

	// Scale triangles
	for(i = 0; i < numTri; i++)
	{
		p1_Writer.Set(i, p1_Writer.Get(i) * scale);
		p2_Writer.Set(i, p2_Writer.Get(i) * scale);
		p3_Writer.Set(i, p3_Writer.Get(i) * scale);
	}

	//copy host vectors into device vectors
	//thrust::copy(p1.begin(), p1.end(), hp1.begin());
	return i;
}

/*int main(int argc, char **argv)
{
	readBinarySTL(argv[1]);
	unsigned int numTri = getNumTriSTL(argv[1]);
	std::cout << numTri << std::endl;
	thrust::device_vector<thrust::tuple<float,float,float>> p1(numTri);
	thrust::device_vector<thrust::tuple<float,float,float>> p2(numTri);
	thrust::device_vector<thrust::tuple<float,float,float>> p3(numTri);
	thrust::device_vector<thrust::tuple<char,char,char>> color(numTri);
	readTriFromBinarySTL(p1, p2, p3, color, argv[1]);
	thrust::host_vector<thrust::tuple<float,float,float>> temp = p1;
	for(int i = 0; i < temp.size(); i++)
		std::cout << temp[i].get<0>() << "\t";
	std::cout << std::endl;
	return 0;
}*/
