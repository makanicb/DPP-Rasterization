#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cassert>
#include <limits>
#include <chrono>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>
#include <thrust/pair.h>
#include <thrust/sequence.h>
#include <thrust/scatter.h>
#include <thrust/scan.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/gather.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

#include <viskores/cont/Algorithm.h>
#include <viskores/cont/ArrayCopy.h>
#include <viskores/cont/ArrayHandle.h>
#include <viskores/cont/ArrayHandleCast.h>
#include <viskores/cont/ArrayHandleConstant.h>
#include <viskores/cont/ArrayHandleCounting.h>
#include <viskores/cont/ArrayHandleDiscard.h>
#include <viskores/cont/ArrayHandlePermutation.h>
#include <viskores/cont/Invoker.h>
#include <viskores/worklet/ScatterCounting.h>
#include <viskores/worklet/ScatterPermutation.h>
#include <viskores/worklet/WorkletMapField.h>

#include "imageWriter.h"
#include "rastByTri.h"

#ifndef DEBUG
#define DEBUG 0 
#endif

struct ExpandWorklet : viskores::worklet::WorkletMapField
{
	using ControlSignature = void (FieldIn input, FieldIn counts, FieldOut output);
	using ExecutionSignature = void(_1, _3, VisitIndex);
	using InputDomain = _1;

	using ScatterType = viskores::worklet::ScatterCounting;
	
	template<typename T>
	VISKORES_EXEC void operator() (const T &in, T &out, viskores::IdComponent visitIndex) const
	{
		out = in;
	}
};

template<typename PermutationStorage>
struct FillImage : viskores::worklet::WorkletMapField
{
	using ControlSignature = void(FieldIn colors, FieldIn map, FieldIn stencil, WholeArrayOut image);
	using ExecutionSignature = void(_1, _2, _3, _4);

	template<typename InputType, typename StencilType, typename PortalType>
	VISKORES_EXEC
	void operator() (const InputType &col, const viskores::Id &pos, const StencilType &sten, PortalType &img) const
	{
		if (sten) 
		{
			img.Set(pos, col);	
		}
	}	

};

__host__ __device__
void getEnds(float x1, float y1, float x2, float y2, float x3, float y3, float y, float &end1, float &end2)
{
		float ed1, ed2, ed3;
		bool e1 = false, e2 = false, e3 = false;
		if((y1 < y2 && y >= y1 && y <= y2) || (y1 > y2 && y >= y2 && y <= y1)) 
		{
			ed1 = (y - y1) * (x2-x1) / (y2-y1) + x1;
			e1 = true;
		}
		if((y2 < y3 && y >= y2 && y <= y3) || (y2 > y3 && y >= y3 && y <= y2)) 
		{
			ed2 = (y - y2) * (x3-x2) / (y3-y2) + x2;
			e2 = true;
		}
		if((y1 < y3 && y >= y1 && y <= y3) || (y1 > y3 && y >= y3 && y <= y1)) 
		{
			ed3 = (y - y1) * (x3-x1) / (y3-y1) + x1;
			e3 = true;
		}

		if(y1 == y2 && y2 == y3 && y3 == y)
		{
			end1 = x1 < x2 ? x1 : x2;
			end1 = end1 < x3 ? end1 : x3;
			end2 = x1 < x2 ? x2 : x1;
			end2 = end2 < x3 ? x3 : end2;
		}
		else if(e1 && e2 && e3)
		{
			end1 = ed1 < ed2 ? ed1 : ed2;
			end1 = end1 < ed3 ? end1 : ed3;
			end2 = ed1 < ed2 ? ed2 : ed1;
			end2 = end2 < ed3 ? ed3 : end2;
		}
		else if (e1 && e2)
		{
			end1 = ed1 < ed2 ? ed1 : ed2;
			end2 = ed1 < ed2 ? ed2 : ed1;
		}
		else if(e2 && e3)
		{
			end1 = ed2 < ed3 ? ed2 : ed3;
			end2 = ed2 < ed3 ? ed3 : ed2;
		}
		else if(e1 && e3)
		{
			end1 = ed1 < ed3 ? ed1 : ed3;
			end2 = ed1 < ed3 ? ed3 : ed1;
		}
}

struct fragCount
{
	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		float x1, y1, x2, y2, x3, y3;
		x1 = thrust::get<0>(thrust::get<0>(t));
		y1 = thrust::get<1>(thrust::get<0>(t));
		x2 = thrust::get<0>(thrust::get<1>(t));
		y2 = thrust::get<1>(thrust::get<1>(t));
		x3 = thrust::get<0>(thrust::get<2>(t));
		y3 = thrust::get<1>(thrust::get<2>(t));
		float minY = y1 < y2 ? y1 : y2;
		minY = minY < y3 ? minY : y3;
		float maxY = y1 > y2 ? y1 : y2;
		maxY = maxY > y3 ? maxY : y3;
		int low = ceil(minY);
		int high = floor(maxY);
		int frags = 0;
		for(int i = low; i <= high; i++)
		{
			float end1, end2;
			getEnds(x1,y1,x2,y2,x3,y3,i,end1,end2);
			frags += floor(end2) - ceil(end1) + 1;
		}

		thrust::get<3>(t) = frags;
	}
};

struct Rasterize : viskores::worklet::WorkletMapField
{
	using ControlSignature = void(
			FieldIn p1,
			FieldIn p2,
			FieldIn p3,
			FieldIn frag_row,
			FieldIn frag_col,
			FieldOut pos,
			FieldOut depth);
	using ExecutionSignature = void(_1, _2, _3, _4, _5, _6, _7);
	template <typename PointType, typename RowColType, typename PositionType, typename DepthType>
	VISKORES_EXEC
	void operator()(const PointType &p1, const PointType &p2, const PointType &p3,
			const RowColType &frag_row, const RowColType &frag_col,
			PositionType &pos, DepthType &depth) const
	{
		float x1, y1, z1, x2, y2, z2, x3, y3, z3;
		x1 = thrust::get<0>(p1);
		y1 = thrust::get<1>(p1);
		z1 = thrust::get<2>(p1);
		x2 = thrust::get<0>(p2);
		y2 = thrust::get<1>(p2);
		z2 = thrust::get<2>(p2);
		x3 = thrust::get<0>(p3);
		y3 = thrust::get<1>(p3);
		z3 = thrust::get<2>(p3);
		/*std::cout << x1 << ","
			  << y1 << ","
			  << z1 << std::endl;
		std::cout << x2 << ","
			  << y2 << ","
			  << z2 << std::endl;
		std::cout << x3 << ","
			  << y3 << ","
			  << z3 << std::endl;*/
		//calculate triangle plane
		float minY = y1 < y2 ? y1 : y2;
		minY = minY < y3 ? minY : y3;
		float y = ceil(minY) + frag_row;
		float end1, end2;
		getEnds(x1,y1,x2,y2,x3,y3,y,end1,end2);
		int x = ceil(end1) + frag_col;
		float z;
		float x_coe = ((y2-y1)*(z3-z1)-(y3-y1)*(z2-z1));
		float y_coe = ((x2-x1)*(z3-z1)-(x3-x1)*(z2-z1));
		float z_coe = ((x2-x1)*(y3-y1)-(x3-x1)*(y2-y1));
		//z_coe is zero if and only if (x2-x1)*(y3-y1)==(x3-x1)*(y2-y1)
		//Then if z_coe is zero then the triangle is a line on the xy plane
		if(z_coe){
			z = z1 - (x_coe*(x-x1)+y_coe*(y1-y))/z_coe;
		}else if(y1 != y2 || y2 != y3 || y1 != y3){
			float minZ, maxZ; 
			getEnds(z1,y1,z2,y2,z3,y3,y,minZ,maxZ);
			z = maxZ;
		}else if(x1 != x2 || x2 != x3 || x1 != x3){
			float minZ, maxZ;
			getEnds(z1,x1,z2,x2,z3,x3,x,minZ,maxZ);
			z = maxZ;
		}else{
			z = z1 > z2 ? z1 : z2;
			z = z > z3 ? z : z3;
		}
		pos = thrust::make_pair(x, y);
		depth = z;
	}
};

struct rowCount
{
	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		float y1, y2, y3;
		y1 = thrust::get<1>(thrust::get<0>(t));
		y2 = thrust::get<1>(thrust::get<1>(t));
		y3 = thrust::get<1>(thrust::get<2>(t)); 
		float minY = y1 < y2 ? y1 : y2; 
		minY = minY < y3 ? minY : y3;
		float maxY = y1 > y2 ? y1 : y2; 
		maxY = maxY > y3 ? maxY : y3;
		thrust::get<3>(t) = floor(maxY) - ceil(minY) + 1; 
	}
};

struct colCount
{
	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		float x1, y1, x2, y2, x3, y3;
		x1 = thrust::get<0>(thrust::get<0>(t));
		y1 = thrust::get<1>(thrust::get<0>(t));
		x2 = thrust::get<0>(thrust::get<1>(t));
		y2 = thrust::get<1>(thrust::get<1>(t));
		x3 = thrust::get<0>(thrust::get<2>(t));
		y3 = thrust::get<1>(thrust::get<2>(t));
		int row = thrust::get<3>(t);
		float minY = y1 < y2 ? y1 : y2;
		minY = minY < y3 ? minY : y3;
		float y = ceil(minY) + row;
		float end1, end2;
		getEnds(x1,y1,x2,y2,x3,y3,y,end1,end2);
		thrust::get<4>(t) = floor(end2) - ceil(end1) + 1;
	}
};

/*
    Given some list of groups let pred be the
    number of elements in each group, and offset
    be the starting position of each group in a
    list of all elements in a supergroup containing
    all groups. expand_int generates a list of all elements
    of the supergroup where the value at an elements indice
    is the index of the group it belongs to.
*/
void expand_int
	(thrust::device_vector<int>::iterator map,
	 thrust::device_vector<int>::iterator pred,
	 thrust::device_vector<int>::iterator start,
	 thrust::device_vector<int>::iterator end,
	 int num)
{
	thrust::scatter_if
		(thrust::counting_iterator<int>(0),
		 thrust::counting_iterator<int>(num),
		 map,
		 pred,
		 start);

	thrust::inclusive_scan
		(start,
		 end,
		 start,
		 thrust::maximum<int>());
}	

/*
   Take a list of values and a list of counts,
   and duplicate each value a number of times
   equal to the count at its index
*/
template<typename T, typename CountT>
void vduplicate(const viskores::cont::ArrayHandle<T> &values,
		 const viskores::cont::ArrayHandle<CountT> &count,
		 viskores::cont::ArrayHandle<T> &output)
{
	viskores::cont::Invoker invoke;
	viskores::worklet::ScatterCounting scatter(count);
	ExpandWorklet expand_worklet;
	invoke(
		expand_worklet,
		scatter,
		values,
		count,
		output
	);

}

/*
    Given some list of groups let counts be the
    number of elements in each group, and assume
    that the order in which groups appear in counts
    is the same as the order in which they appear in
    a supergroup containing all groups. vexpand generates
    a list of all elements of the supergroup where
    the value at an element's indice is the index 
    of the group it belongs to.
*/
template<typename T, typename CountT>
void vexpand(viskores::cont::ArrayHandle<CountT> &counts,
		 viskores::cont::ArrayHandle<T> &output)
{
	viskores::Id length = counts.GetNumberOfValues();
	viskores::cont::ArrayHandle<T> sequence;
	viskores::cont::ArrayCopy
		(viskores::cont::make_ArrayHandleCounting<T>(0, 1, length),
		 sequence);
	vduplicate<T, CountT>(
		sequence,
		counts,
		output
	);
}

/*
   Let map be a list associating elements to their
   groups, and src be a list of group offsets.
   index_int generates a list where the value at an
   element's index is its index within its group.
*/
void index_int
	(thrust::device_vector<int>::iterator map,
	 thrust::device_vector<int>::iterator src,
	 thrust::device_vector<int>::iterator out,
	 int num)
{
	thrust::transform
		(thrust::counting_iterator<int>(0),
		 thrust::counting_iterator<int>(num),
		 thrust::make_permutation_iterator(src, map),
		 out,
		 thrust::minus<int>());
}

/*
   Let map be a list associating elements to their
   groups, and src be a list of group offsets.
   vindex generates a list where the value at an
   element's index is its index within its group.
*/
template<typename IndexType, typename ValueType>
void vindex
	(viskores::cont::ArrayHandle<IndexType> &map,
	 viskores::cont::ArrayHandle<ValueType> &src,
	 viskores::cont::ArrayHandle<ValueType> &out)
{
	viskores::Id length = map.GetNumberOfValues();
	viskores::cont::Algorithm::Transform
		(viskores::cont::make_ArrayHandleCounting<ValueType>(0, 1, length),
		 viskores::cont::make_ArrayHandlePermutation(map, src),
		 out, thrust::minus<ValueType>());
}


void print_int_vec(thrust::device_vector<int>::iterator start,
		   thrust::device_vector<int>::iterator end)
{
	for(; start < end; start++)
		std::cout << *start << " ";
	std::cout << std::endl;
}

void print_pair_vec(thrust::device_vector<thrust::pair<int,int>>::iterator start,
		    thrust::device_vector<thrust::pair<int,int>>::iterator end)
{
	for(; start < end; start++)
	{
		thrust::pair<int,int> temp = *start;
		std::cout << temp.first << "," << temp.second << "\t";
	}
	std::cout << std::endl;
}

void print_float_vec(thrust::device_vector<float>::iterator start,
		     thrust::device_vector<float>::iterator end)
{
	for(; start < end; start++)
		std::cout << *start << " ";
	std::cout << std::endl;
}

template<typename T>
void print_ArrayHandle(const viskores::cont::ArrayHandle<T> &arr)
{
	auto arr_Reader = arr.ReadPortal();
	for (viskores::Id i = 0; i < arr_Reader.GetNumberOfValues(); i++)
	{
		std::cout << arr_Reader.Get(i) << "\t";
	}
	std::cout << std::endl;
}

/*
struct key_equality
{
	__host__ __device__	
	bool operator()
		(thrust::pair<thrust::pair<int,int>, int> p1, thrust::pair<thrust::pair<int,int>, int> p2)
	{
		return thrust::get<0>(thrust::get<0>(p1)) == thrust::get<0>(thrust::get<0>(p2)) &&
		       thrust::get<1>(thrust::get<0>(p1)) == thrust::get<1>(thrust::get<0>(p2));
	}
};
*/

struct findPositions
{
	thrust::device_vector<thrust::pair<int,int>>::iterator start;
	thrust::device_vector<thrust::pair<int,int>>::iterator stop;
	findPositions
		(thrust::device_vector<thrust::pair<int,int>>::iterator _start, thrust::device_vector<thrust::pair<int,int>>::iterator _stop)
		: start(_start), stop(_stop) {}

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		thrust::pair<int,int> pos = thrust::get<0>(t);
		thrust::get<1>(t) = (int)(thrust::find(start, stop, pos) - start);
	}
};

struct ToRowMajor : viskores::worklet::WorkletMapField
{
	int w;
	ToRowMajor(int _w) : w(_w) {}

	using ControlSignature = void(FieldIn coordinates, FieldOut indices);
	using ExecutionSignature = _2(_1);

	VISKORES_EXEC
	int operator()(const thrust::pair<int,int> &pos) const
	{
		return pos.first + pos.second * w;
	}
};

void RasterizeTriangles(thrust::device_vector<thrust::tuple<float, float, float>> &p1,
		thrust::device_vector<thrust::tuple<float, float, float>> &p2,
		thrust::device_vector<thrust::tuple<float, float, float>> &p3,
		thrust::device_vector<thrust::tuple<char, char, char>> &color,
		int numTri, int width, int height, Image &final_image)
{
	//Set up timing systems
	thrust::host_vector<std::chrono::time_point<std::chrono::high_resolution_clock>> timer;
	//time: function start
	timer.push_back(std::chrono::high_resolution_clock::now());	
	
	//Define a Viskores Invoker
	viskores::cont::Invoker invoke;

/*
   RASTERIZE
*/

#if DEBUG > 0
	std::cout << "Count fragments" << std::endl;
#endif
#if DEBUG > 1 
	std::cout << numTri << " Triangles" << std::endl;
#endif	
	thrust::device_vector<int> frags(numTri);

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(p1.begin(), p2.begin(), p3.begin(), frags.begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(p1.end(), p2.end(), p3.end(), frags.end())),
			 fragCount());
#if DEBUG > 1 
	std::cout << "# frags by triange: " << std::endl;
	print_int_vec(frags.begin(), frags.end());
#endif

	thrust::device_vector<int> write_index(numTri);

	thrust::exclusive_scan(frags.begin(), frags.end(), write_index.begin());
#if DEBUG > 1
	std::cout << "write position by triange: " << std::endl;
	print_int_vec(write_index.begin(), write_index.end());
#endif

	int fragments = write_index[numTri-1] + frags[numTri-1];
#if DEBUG > 1	
	std::cout << "Number of fragments: " << fragments << std::endl;
#endif
#if DEBUG > 0
	std::cout << "Get fragments" << std::endl;
#endif

	thrust::device_vector<int> frag_tri(fragments);
	expand_int(write_index.begin(), frags.begin(), frag_tri.begin(), frag_tri.end(), numTri);
#if DEBUG > 3
	std::cout << "Which triangle does each fragment belong to?" << std::endl;
	print_int_vec(frag_tri.begin(), frag_tri.end());
#endif
/*
	thrust::scatter_if
		(thrust::counting_iterator<int>(0),
		 thrust::counting_iterator<int>(2),
		 write_index.begin(),
		 frags.begin(),
		 frag_pos.begin());

	thrust::inclusive_scan
		(frag_pos.begin(),
		 frag_pos.end(),
		 frag_pos.begin(),
		 thrust::maximum<int>());
	
	thrust::device_vector<int> frag_ind(fragments);

	thrust::transform
		(thrust::counting_iterator<int>(0),
		 thrust::counting_iterator<int>(fragments),
		 thrust::make_permutation_iterator(write_index.begin(), frag_pos.begin()), frag_ind.begin(),
		 thrust::minus<int>());
*/	
	thrust::device_vector<int> rows(numTri);
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(p1.begin(), p2.begin(), p3.begin(), rows.begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(p1.end(), p2.end(), p3.end(), rows.end())),
			 rowCount());
#if DEBUG > 1
	std::cout << "How many rows does each triangle have?" << std::endl;
	for(int i = 0; i < numTri; i++)
		std::cout << rows[i] << " ";
	std::cout << std::endl;
#endif

	thrust::device_vector<int> row_off(numTri);
	thrust::exclusive_scan(rows.begin(), rows.end(), row_off.begin());
#if DEBUG > 1
	std::cout << "What is the row offset of each triangle?" << std::endl;
	for(int i = 0; i < numTri; i++)
		std::cout << row_off[i] << " ";
	std::cout << std::endl;
#endif

	int num_rows = row_off[numTri-1] + rows[numTri-1];

	thrust::device_vector<int> tri_ptr(num_rows);
	
	expand_int(row_off.begin(), rows.begin(), tri_ptr.begin(), tri_ptr.end(), numTri);
#if DEBUG > 2 
	std::cout << "What triangle does each row belong to?" << std::endl;
	for(int i = 0; i < num_rows; i++)
		std::cout << tri_ptr[i] << " ";
	std::cout << std::endl;	
#endif

	thrust::device_vector<int> row_ptr(num_rows);

	index_int(tri_ptr.begin(), row_off.begin(), row_ptr.begin(), num_rows);
#if DEBUG > 2 
	std::cout << "The index of each row." << std::endl;
	print_int_vec(row_ptr.begin(), row_ptr.end());
#endif

	thrust::device_vector<int> col_count(num_rows);

	thrust::for_each
		(thrust::make_zip_iterator(thrust::make_tuple
		 	(thrust::make_permutation_iterator(p1.begin(), tri_ptr.begin()),
			 thrust::make_permutation_iterator(p2.begin(), tri_ptr.begin()),
			 thrust::make_permutation_iterator(p3.begin(), tri_ptr.begin()),
			 row_ptr.begin(),
			 col_count.begin())),
		thrust::make_zip_iterator(thrust::make_tuple
		 	(thrust::make_permutation_iterator(p1.begin(), tri_ptr.end()),
			 thrust::make_permutation_iterator(p2.begin(), tri_ptr.end()),
			 thrust::make_permutation_iterator(p3.begin(), tri_ptr.end()),
			 row_ptr.end(),
			 col_count.end())),
		colCount());
#if DEBUG > 2
	std::cout << "How many columns does each row have?" << std::endl;
	print_int_vec(col_count.begin(), col_count.end());
#endif
	//Copy vectors to ArrayHandles
	viskores::cont::ArrayHandle<int> vcol_count = 
		viskores::cont::make_ArrayHandle(thrust::raw_pointer_cast(col_count.data()), col_count.size(), viskores::CopyFlag::On);

	viskores::cont::ArrayHandle<viskores::Id> vcol_off;

	viskores::cont::Algorithm::ScanExclusive
		(viskores::cont::make_ArrayHandleCast<viskores::Id>(vcol_count), vcol_off);
#if DEBUG > 2 
	std::cout << "Column offsets by row" << std::endl;
	print_int_vec(col_off.begin(), col_off.end());
	std::cout << "Number of columns " <<  col_off[num_rows-1] + col_count[num_rows-1] << std::endl;
#endif
	assert((fragments == (int)vcol_off.ReadPortal().Get(num_rows-1) + (int)vcol_count.ReadPortal().Get(num_rows-1)));
	//Copy vectors to array handles
	std::vector<viskores::Id> tmp_frag_tri(frag_tri.begin(), frag_tri.end());
	viskores::cont::ArrayHandle<viskores::Id> vfrag_tri = 
		viskores::cont::make_ArrayHandle(tmp_frag_tri, viskores::CopyFlag::On);
	std::vector<viskores::Id> tmp_row_off(row_off.begin(), row_off.end());
	viskores::cont::ArrayHandle<viskores::Id> vrow_off = 
		viskores::cont::make_ArrayHandle(tmp_row_off, viskores::CopyFlag::On);

	//Initialize ArrayHandles
	viskores::cont::ArrayHandle<viskores::Id> vfrag_row;
	viskores::cont::ArrayHandle<viskores::Id> vfrag_col;

	//Determine fragment rows and columns
	vexpand(vcol_count, vfrag_row);
	std::cout << "Frag Rows" << std::endl;
	//print_ArrayHandle(vfrag_row);

	//temporary copies
	/*
	viskores::cont::ArrayHandle<viskores::Id> vtmp_frag_row;
	viskores::cont::ArrayHandle<viskores::Id> vtmp_frag_col;
	vtmp_frag_row.DeepCopyFrom(vfrag_row);
	vtmp_frag_col.DeepCopyFrom(vfrag_col);
	*/

	vindex(vfrag_row, vcol_off, vfrag_col);

	viskores::cont::Algorithm::Transform
		(vfrag_row,
		 viskores::cont::make_ArrayHandlePermutation(vfrag_tri, vrow_off),
		 vfrag_row,
		 thrust::minus<viskores::Id>());
	//std::cout << "Size of frag_row, frag_col: " <<
	//	vfrag_row.GetNumberOfValues() << ", " <<
	//	vfrag_col.GetNumberOfValues() << std::endl;
	//std::cout << "Frag Col" << std::endl;
	//print_ArrayHandle(vfrag_col);
		 
#if DEBUG > 3 
	std::cout << "Frag positions by row and column in every triangle." << std::endl;
	print_int_vec(frag_row.begin(), frag_row.end());
	print_int_vec(frag_col.begin(), frag_col.end());
#endif
	//Copy vectors to ArrayHandles
	viskores::cont::ArrayHandle<thrust::tuple<float,float,float>> vp1 = 
		viskores::cont::make_ArrayHandle(thrust::raw_pointer_cast(p1.data()), p1.size(), viskores::CopyFlag::On);
	viskores::cont::ArrayHandle<thrust::tuple<float,float,float>> vp2 = 
		viskores::cont::make_ArrayHandle(thrust::raw_pointer_cast(p2.data()), p2.size(), viskores::CopyFlag::On);
	viskores::cont::ArrayHandle<thrust::tuple<float,float,float>> vp3 = 
		viskores::cont::make_ArrayHandle(thrust::raw_pointer_cast(p3.data()), p3.size(), viskores::CopyFlag::On);

	//Initialize ArrayHandles
	viskores::cont::ArrayHandle<thrust::pair<int,int>> vpos;
	viskores::cont::ArrayHandle<float> vdepth;

	/*thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				thrust::make_permutation_iterator(p1.begin(), frag_tri.begin()),
				thrust::make_permutation_iterator(p2.begin(), frag_tri.begin()),
				thrust::make_permutation_iterator(p3.begin(), frag_tri.begin()),
				frag_row.begin(), frag_col.begin(), pos.begin(), depth.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				thrust::make_permutation_iterator(p1.begin(), frag_tri.end()),
				thrust::make_permutation_iterator(p2.begin(), frag_tri.end()),
				thrust::make_permutation_iterator(p3.begin(), frag_tri.end()),
				frag_row.end(), frag_col.end(), pos.end(), depth.end())),
		rasterize());*/

	//Rasterize
	Rasterize rasterize;	
	invoke(rasterize,
		viskores::cont::make_ArrayHandlePermutation(vfrag_tri, vp1),
		viskores::cont::make_ArrayHandlePermutation(vfrag_tri, vp2),
		viskores::cont::make_ArrayHandlePermutation(vfrag_tri, vp3),
		vfrag_row, vfrag_col, vpos, vdepth);

	/*auto tmp_pos_Reader = vpos.ReadPortal();
	for(viskores::Id i = 0; i < tmp_pos_Reader.GetNumberOfValues(); i++)
	{
		std::cout << "(" << thrust::get<0>(tmp_pos_Reader.Get(i)) << ", " << 
			thrust::get<1>(tmp_pos_Reader.Get(i)) << ")\t";
	}
	std::cout << std::endl;
	auto tmp_dep_Reader = vdepth.ReadPortal();
	for(viskores::Id i = 0; i < tmp_dep_Reader.GetNumberOfValues(); i++)
	{
		std::cout << tmp_dep_Reader.Get(i) << "\t";
	}
	std::cout << std::endl;*/

#if DEBUG > 3
	std::cout << "Position and depth of fragments" << std::endl;
	print_pair_vec(pos.begin(), pos.end());
	print_float_vec(depth.begin(), depth.end());
#endif
	//Copy vectors to ArrayHandles
	viskores::cont::ArrayHandle<thrust::tuple<char,char,char>> vcolor = 
		viskores::cont::make_ArrayHandle(thrust::raw_pointer_cast(color.data()), color.size(), viskores::CopyFlag::On);

	//Gather the color of each fragment
	viskores::cont::ArrayHandlePermutation<viskores::cont::ArrayHandle<viskores::Id>, viskores::cont::ArrayHandle<thrust::tuple<char,char,char>>> vfrag_colors(vfrag_tri, vcolor);

	//time: rasterized triangles. acquired all fragments
	timer.push_back(std::chrono::high_resolution_clock::now());

/*
   SORT
*/

#if DEBUG > 0	
	std::cout << "find fragments to write" << std::endl;

	std::cout << "\tcopy position" << std::endl;
#endif

	//Allocate ArrayHandles for Sorting
	viskores::cont::ArrayHandle<thrust::pair<int, int>> vcpos;
	vcpos.DeepCopyFrom(vpos);	
	viskores::cont::ArrayHandleCounting<viskores::Id> tmp_inds(0, 1, fragments);
	viskores::cont::ArrayHandle<viskores::Id> vsorted_inds;
	viskores::cont::Algorithm::Copy(tmp_inds, vsorted_inds);

#if DEBUG > 0
	std::cout << "\tsort fragments" << std::endl;
#endif
	viskores::cont::Algorithm::SortByKey(vcpos, vsorted_inds);
	viskores::cont::ArrayHandlePermutation<viskores::cont::ArrayHandle<viskores::Id>, viskores::cont::ArrayHandlePermutation<viskores::cont::ArrayHandle<viskores::Id>, viskores::cont::ArrayHandle<thrust::tuple<char,char,char>>>> vcfrag_colors(vsorted_inds, vfrag_colors);
	viskores::cont::ArrayHandlePermutation<viskores::cont::ArrayHandle<viskores::Id>, viskores::cont::ArrayHandle<float>> vcdepth(vsorted_inds, vdepth);

#if DEBUG > 3
	std::cout << "Sorted" << std::endl;
	print_pair_vec(cpos.begin(), cpos.end());
	print_int_vec(sorted_inds.begin(), sorted_inds.end());
	print_float_vec(cdepth.begin(), cdepth.end());
#endif
	//time: sorted fragments
	timer.push_back(std::chrono::high_resolution_clock::now());

/*
   SELECT
*/

#if DEBUG > 0
	std::cout << "\tget fragments at lowest depth" << std::endl;
#endif
/*
	//count the number of unique positions
	int unique_positions;
	{
		viskores::cont::ArrayHandle<thrust::pair<int,int>> tmp_pos;
		viskores::cont::Algorithm::Copy(vcpos, tmp_pos);
		viskores::cont::Algorithm::Unique(tmp_pos);
		unique_positions = tmp_pos.GetNumberOfValues();
	}
#if DEBUG > 1
	std::cout << "\tunique positions = " << unique_positions << std::endl;
#endif
*/
	viskores::cont::ArrayHandle<thrust::pair<int,int>> vtrue_fragments;
	viskores::cont::ArrayHandle<float> vmin_depth;
	viskores::cont::ArrayHandle<int> vpos_count;
	viskores::cont::Algorithm::ReduceByKey(vcpos, vcdepth, vtrue_fragments, vmin_depth, thrust::maximum<float>());
	viskores::cont::Algorithm::ReduceByKey(vcpos, viskores::cont::make_ArrayHandleConstant<int>(1, fragments),
		       vtrue_fragments, vpos_count, thrust::plus<int>());	

#if DEBUG > 3
	std::cout << "Number of duplicates at each unique position" << std::endl;
	print_int_vec(pos_count.begin(), pos_count.end());
#endif
#if DEBUG > 0
	std::cout << "\tGet the minimum depth of each unique position" << std::endl;
#endif
	/* Thrust Implementation

	thrust::device_vector<int> pos_start_ind(unique_positions);
	thrust::exclusive_scan(pos_count.begin(), pos_count.end(), pos_start_ind.begin());
#if DEBUG > 3
	std::cout << "Offset by unique position" << std::endl;
	print_int_vec(pos_start_ind.begin(), pos_start_ind.end());
#endif
	thrust::device_vector<int> depth_map(fragments);
	expand_int(pos_start_ind.begin(), pos_count.begin(), depth_map.begin(), depth_map.end(), unique_positions);
#if DEBUG > 3
	std::cout << "Min depth gather position by fragment" << std::endl;
	print_int_vec(depth_map.begin(), depth_map.end());
#endif
	thrust::device_vector<float> exp_min_depth(fragments);
	thrust::gather(depth_map.begin(), depth_map.end(), min_depth.begin(), exp_min_depth.begin());

	*/

	/* Viskores Implementation */

	viskores::cont::ArrayHandle<float> vexp_min_depth;
	vduplicate(vmin_depth, vpos_count, vexp_min_depth);

#if DEBUG > 3
	std::cout << "Min depth by fragment" << std::endl;
	print_float_vec(exp_min_depth.begin(), exp_min_depth.end());
#endif
/*
	//std::cout << "Min depth" << std::endl;
	//print_pair_vec(true_fragments.begin(), true_fragments.end());
	//print_float_vec(min_depth.begin(), min_depth.end());

	//thrust::device_vector<thrust::pair<int,int>>::iterator true_end = thrust::unique(true_fragments.begin(), true_fragments.end()) - 1;
	//print_pair_vec(true_fragments.begin(), true_fragments.end());

	std::cout << "\tfor each position, get the shallowest depth of a fragment at that position" << std::endl;
	thrust::device_vector<int> find_real(fragments);
	std::cout <<"\t\tfind each fragment position in list of lowest fragment positions" << std::endl;
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(pos.begin(), find_real.begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(pos.end(), find_real.end())),
			 findPositions(true_fragments.begin(), true_fragments.end()));
	//print_int_vec(find_real.begin(), find_real.end());
	std::cout << "\t\tgather the shallowest depth for each fragment position" << std::endl;
	thrust::device_vector<float> min_depth_by_fragment(fragments);
	thrust::gather(find_real.begin(), find_real.end(), min_depth.begin(), min_depth_by_fragment.begin());
*/
#if DEBUG > 3
	std::cout << "Min Depth at fragment position vs fragment depth" << std::endl;
	print_float_vec(exp_min_depth.begin(), exp_min_depth.end());
	print_float_vec(cdepth.begin(), cdepth.end());
#endif
#if DEBUG > 0
	std::cout << "\tchoose fragments to write" << std::endl;
#endif
	viskores::cont::ArrayHandle<bool> vwrite_frag;
	viskores::cont::Algorithm::Transform(vexp_min_depth, vcdepth, vwrite_frag, thrust::equal_to<float>());

	/*
	//Convert ArrayHandles to Thrust vectors

	//Create portals for reading
	auto pos_Reader = vcpos.ReadPortal();
	auto depth_Reader = vcdepth.ReadPortal();
	auto color_Reader = vcfrag_colors.ReadPortal();
	auto inds_Reader = vsorted_inds.ReadPortal();

	auto true_frag_reader = vtrue_fragments.ReadPortal();
	auto min_depth_reader = vmin_depth.ReadPortal();
	auto pos_count_reader = vpos_count.ReadPortal();
	auto exp_min_depth_reader = vexp_min_depth.ReadPortal();
	auto write_frag_reader = vwrite_frag.ReadPortal();

	//Create Thrust vectors
	thrust::device_vector<thrust::pair<int,int>> cpos(
		viskores::cont::ArrayPortalToIteratorBegin(pos_Reader),
		viskores::cont::ArrayPortalToIteratorEnd(pos_Reader)
	);
	thrust::device_vector<float> cdepth(
		viskores::cont::ArrayPortalToIteratorBegin(depth_Reader),
		viskores::cont::ArrayPortalToIteratorEnd(depth_Reader)
	);
	thrust::device_vector<thrust::tuple<char,char,char>> cfrag_colors(
		viskores::cont::ArrayPortalToIteratorBegin(color_Reader),
		viskores::cont::ArrayPortalToIteratorEnd(color_Reader)
	);
	thrust::device_vector<int> sorted_inds(
		viskores::cont::ArrayPortalToIteratorBegin(inds_Reader),
		viskores::cont::ArrayPortalToIteratorEnd(inds_Reader)
	);

	thrust::device_vector<thrust::pair<int,int>> true_fragments(
		viskores::cont::ArrayPortalToIteratorBegin(true_frag_reader),
		viskores::cont::ArrayPortalToIteratorEnd(true_frag_reader)
	);
	thrust::device_vector<float> min_depth(
		viskores::cont::ArrayPortalToIteratorBegin(min_depth_reader),
		viskores::cont::ArrayPortalToIteratorEnd(min_depth_reader)
	);
	thrust::device_vector<int> pos_count(
		viskores::cont::ArrayPortalToIteratorBegin(pos_count_reader),
		viskores::cont::ArrayPortalToIteratorEnd(pos_count_reader)
	);
	thrust::device_vector<float> exp_min_depth(
		viskores::cont::ArrayPortalToIteratorBegin(exp_min_depth_reader),
		viskores::cont::ArrayPortalToIteratorEnd(exp_min_depth_reader)
	);
	thrust::device_vector<bool> write_frag(
		viskores::cont::ArrayPortalToIteratorBegin(write_frag_reader),
		viskores::cont::ArrayPortalToIteratorEnd(write_frag_reader)
	);
	*/

#if DEBUG > 3
	std::cout << "Write fragment?" << std::endl;
	for(int i = 0; i < fragments; i++)
		std::cout << write_frag[i] << " ";
	std::cout << std::endl;
#endif
	//time: got visible fragments
	timer.push_back(std::chrono::high_resolution_clock::now());

/*
   WRITE
*/

#if DEBUG > 0
	std::cout << "write fragments" << std::endl;
#endif

	viskores::cont::ArrayHandle<viskores::Id> vrowMajorPos;
	ToRowMajor to_row_major(width);
	invoke(to_row_major, vcpos, vrowMajorPos);

#if DEBUG > 3
	std::cout << "Row major position by fragment" << std::endl;
	print_int_vec(rowMajorPos.begin(), rowMajorPos.end());
#endif

	//viskores::cont::ArrayHandle<thrust::tuple<char,char,char>> vbg;
	//vbg.AllocateAndFill(width * height, thrust::make_tuple<char,char,char>(127,127,127));
	viskores::cont::ArrayHandle<thrust::tuple<char,char,char>> vimg;
	vimg.AllocateAndFill(width * height, thrust::make_tuple<char,char,char>(127,127,127));
	/*
	std::cout << vcfrag_colors.GetNumberOfValues() << std::endl;
	std::cout << vrowMajorPos.GetNumberOfValues() << std::endl;
	std::cout << vwrite_frag.GetNumberOfValues() << std::endl;
	std::cout << vimg.GetNumberOfValues() << std::endl;
	*/
	//auto max_pos = viskores::cont::Algorithm::Reduce(vrowMajorPos, (viskores::Id) 0,
	//	       [](const auto& a, const auto& b){return std::max(a,b);});	
	//std::cout << max_pos << std::endl;
	FillImage<viskores::cont::StorageTagBasic> fill_image;
	invoke(
		fill_image,
		vcfrag_colors,
		vrowMajorPos,
		vwrite_frag,
		vimg
	);

	auto img_Reader = vimg.ReadPortal();
	int count = 0;
	for(viskores::Id i = 0; i < img_Reader.GetNumberOfValues(); i++)
	{
		thrust::tuple<char,char,char> t = img_Reader.Get(i);
		final_image.data[count++] = thrust::get<0>(t);
		final_image.data[count++] = thrust::get<1>(t);
		final_image.data[count++] = thrust::get<2>(t);
	}
	//time: write final image to output
	timer.push_back(std::chrono::high_resolution_clock::now());

/*
   DONE
*/

	//char *col = final_image.data;
	//for(int i = 0; i < 60; i+=3)
	//{
	//	std::cout<<(int)col[i]<<","<<(int)col[i+1]<<","<<(int)col[i+2]<<std::endl;
	//}
	auto p = timer.begin();
	for(auto i = timer.begin() + 1; i != timer.end(); i++)
	{
		auto duration = std::chrono::duration_cast<std::chrono::microseconds>(*i - *p);
		p = i;
		std::cout << "\t" << duration.count();	
	}
	std::cout << std::endl;
}
