#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cassert>
#include <limits>
#include <chrono>

/*
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>
#include <thrust/pair.h>
#include <thrust/sequence.h>
#include <thrust/scatter.h>
#include <thrust/scan.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/gather.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
*/

#include <viskores/cont/Algorithm.h>
#include <viskores/cont/ArrayCopy.h>
#include <viskores/cont/ArrayHandle.h>
#include <viskores/cont/ArrayHandleCast.h>
#include <viskores/cont/ArrayHandleConstant.h>
#include <viskores/cont/ArrayHandleCounting.h>
#include <viskores/cont/ArrayHandleDiscard.h>
#include <viskores/cont/ArrayHandlePermutation.h>
#include <viskores/cont/Invoker.h>
#include <viskores/Types.h>
#include <viskores/worklet/ScatterCounting.h>
#include <viskores/worklet/ScatterPermutation.h>
#include <viskores/worklet/WorkletMapField.h>

#include "imageWriter.h"
#include "rastByTri.h"

#ifndef DEBUG
#define DEBUG 0 
#endif

template<typename T>
struct my_maximum
{
	T operator()(const T &lhs, const T &rhs) const
	{
		return lhs > rhs ? lhs : rhs;
	}
};

struct ExpandWorklet : viskores::worklet::WorkletMapField
{
	using ControlSignature = void (FieldIn input, FieldIn counts, FieldOut output);
	using ExecutionSignature = void(_1, _3, VisitIndex);
	using InputDomain = _1;

	using ScatterType = viskores::worklet::ScatterCounting;
	
	template<typename T>
	VISKORES_EXEC void operator() (const T &in, T &out, viskores::IdComponent visitIndex) const
	{
		out = in;
	}
};

template<typename PermutationStorage>
struct FillImage : viskores::worklet::WorkletMapField
{
	using ControlSignature = void(FieldIn colors, FieldIn map, FieldIn stencil, WholeArrayOut image);
	using ExecutionSignature = void(_1, _2, _3, _4);

	template<typename InputType, typename StencilType, typename PortalType>
	VISKORES_EXEC
	void operator() (const InputType &col, const viskores::Id &pos, const StencilType &sten, PortalType &img) const
	{
		if (sten) 
		{
			img.Set(pos, col);	
		}
	}	

};

VISKORES_EXEC
void getEnds(float x1, float y1, float x2, float y2, float x3, float y3, float y, float &end1, float &end2)
{
		float ed1, ed2, ed3;
		bool e1 = false, e2 = false, e3 = false;
		if((y1 < y2 && y >= y1 && y <= y2) || (y1 > y2 && y >= y2 && y <= y1)) 
		{
			ed1 = (y - y1) * (x2-x1) / (y2-y1) + x1;
			e1 = true;
		}
		if((y2 < y3 && y >= y2 && y <= y3) || (y2 > y3 && y >= y3 && y <= y2)) 
		{
			ed2 = (y - y2) * (x3-x2) / (y3-y2) + x2;
			e2 = true;
		}
		if((y1 < y3 && y >= y1 && y <= y3) || (y1 > y3 && y >= y3 && y <= y1)) 
		{
			ed3 = (y - y1) * (x3-x1) / (y3-y1) + x1;
			e3 = true;
		}

		if(y1 == y2 && y2 == y3 && y3 == y)
		{
			end1 = x1 < x2 ? x1 : x2;
			end1 = end1 < x3 ? end1 : x3;
			end2 = x1 < x2 ? x2 : x1;
			end2 = end2 < x3 ? x3 : end2;
		}
		else if(e1 && e2 && e3)
		{
			end1 = ed1 < ed2 ? ed1 : ed2;
			end1 = end1 < ed3 ? end1 : ed3;
			end2 = ed1 < ed2 ? ed2 : ed1;
			end2 = end2 < ed3 ? ed3 : end2;
		}
		else if (e1 && e2)
		{
			end1 = ed1 < ed2 ? ed1 : ed2;
			end2 = ed1 < ed2 ? ed2 : ed1;
		}
		else if(e2 && e3)
		{
			end1 = ed2 < ed3 ? ed2 : ed3;
			end2 = ed2 < ed3 ? ed3 : ed2;
		}
		else if(e1 && e3)
		{
			end1 = ed1 < ed3 ? ed1 : ed3;
			end2 = ed1 < ed3 ? ed3 : ed1;
		}
}

struct FragCount : viskores::worklet::WorkletMapField
{
	using ControlSignature = void(
		FieldIn p1,
		FieldIn p2,
		FieldIn p3,
		FieldOut fragCount);
	using ExecutionSignature = void(_1, _2, _3, _4);
	template <typename PointType, typename FragCountType>
	VISKORES_EXEC
	void operator()(const PointType &p1, const PointType &p2, const PointType &p3,
			FragCountType &fragCount) const
	{
		float x1, y1, x2, y2, x3, y3;
		x1 = p1[0];
		y1 = p1[1];
		x2 = p2[0];
		y2 = p2[1];
		x3 = p3[0];
		y3 = p3[1];
		float minY = y1 < y2 ? y1 : y2;
		minY = minY < y3 ? minY : y3;
		float maxY = y1 > y2 ? y1 : y2;
		maxY = maxY > y3 ? maxY : y3;
		int low = ceil(minY);
		int high = floor(maxY);
		int frags = 0;
		for(int i = low; i <= high; i++)
		{
			float end1, end2;
			getEnds(x1,y1,x2,y2,x3,y3,i,end1,end2);
			frags += floor(end2) - ceil(end1) + 1;
		}

		fragCount = frags;
	}
};

struct Rasterize : viskores::worklet::WorkletMapField
{
	using ControlSignature = void(
			FieldIn p1,
			FieldIn p2,
			FieldIn p3,
			FieldIn frag_row,
			FieldIn frag_col,
			FieldOut pos,
			FieldOut depth);
	using ExecutionSignature = void(_1, _2, _3, _4, _5, _6, _7);
	template <typename PointType, typename RowColType, typename PositionType, typename DepthType>
	VISKORES_EXEC
	void operator()(const PointType &p1, const PointType &p2, const PointType &p3,
			const RowColType &frag_row, const RowColType &frag_col,
			PositionType &pos, DepthType &depth) const
	{
		float x1, y1, z1, x2, y2, z2, x3, y3, z3;
		x1 = p1[0];
		y1 = p1[1];
		z1 = p1[2];
		x2 = p2[0];
		y2 = p2[1];
		z2 = p2[2];
		x3 = p3[0];
		y3 = p3[1];
		z3 = p3[2];
		/*std::cout << x1 << ","
			  << y1 << ","
			  << z1 << std::endl;
		std::cout << x2 << ","
			  << y2 << ","
			  << z2 << std::endl;
		std::cout << x3 << ","
			  << y3 << ","
			  << z3 << std::endl;*/
		//calculate triangle plane
		float minY = y1 < y2 ? y1 : y2;
		minY = minY < y3 ? minY : y3;
		float y = ceil(minY) + frag_row;
		float end1, end2;
		getEnds(x1,y1,x2,y2,x3,y3,y,end1,end2);
		int x = ceil(end1) + frag_col;
		float z;
		float x_coe = ((y2-y1)*(z3-z1)-(y3-y1)*(z2-z1));
		float y_coe = ((x2-x1)*(z3-z1)-(x3-x1)*(z2-z1));
		float z_coe = ((x2-x1)*(y3-y1)-(x3-x1)*(y2-y1));
		//z_coe is zero if and only if (x2-x1)*(y3-y1)==(x3-x1)*(y2-y1)
		//Then if z_coe is zero then the triangle is a line on the xy plane
		if(z_coe){
			z = z1 - (x_coe*(x-x1)+y_coe*(y1-y))/z_coe;
		}else if(y1 != y2 || y2 != y3 || y1 != y3){
			float minZ, maxZ; 
			getEnds(z1,y1,z2,y2,z3,y3,y,minZ,maxZ);
			z = maxZ;
		}else if(x1 != x2 || x2 != x3 || x1 != x3){
			float minZ, maxZ;
			getEnds(z1,x1,z2,x2,z3,x3,x,minZ,maxZ);
			z = maxZ;
		}else{
			z = z1 > z2 ? z1 : z2;
			z = z > z3 ? z : z3;
		}
		pos = viskores::make_Vec(x, y);
		depth = z;
	}
};

struct RowCount : viskores::worklet::WorkletMapField
{
	using ControlSignature = void(
		FieldIn p1,
		FieldIn p2,
		FieldIn p3,
		FieldOut rowCount
	);
	using ExecutionSignature = void(_1, _2, _3, _4);
	template <typename PointType, typename RowCountType>
	VISKORES_EXEC
	void operator()(const PointType &p1, const PointType &p2, const PointType &p3,
			RowCountType &rowCount) const
	{
		float y1, y2, y3;
		y1 = p1[1];
		y2 = p2[1];
		y3 = p3[1]; 
		float minY = y1 < y2 ? y1 : y2; 
		minY = minY < y3 ? minY : y3;
		float maxY = y1 > y2 ? y1 : y2; 
		maxY = maxY > y3 ? maxY : y3;
		rowCount = floor(maxY) - ceil(minY) + 1; 
	}
};

struct ColCount : viskores::worklet::WorkletMapField
{
	using ControlSignature = void(
		FieldIn p1,
		FieldIn p2,
		FieldIn p3,
		FieldIn row,
		FieldOut colCount);
	using ExecutionSignature = void(_1, _2, _3, _4, _5);

	template <typename PointType, typename RowType, typename ColCountType>
	VISKORES_EXEC
	void operator()(const PointType &p1, const PointType &p2, const PointType &p3,
			const RowType &row, ColCountType &colCount) const
	{
		float x1, y1, x2, y2, x3, y3;
		x1 = p1[0];
		y1 = p1[1];
		x2 = p2[0];
		y2 = p2[1];
		x3 = p3[0];
		y3 = p3[1];
		float minY = y1 < y2 ? y1 : y2;
		minY = minY < y3 ? minY : y3;
		float y = ceil(minY) + row;
		float end1, end2;
		getEnds(x1,y1,x2,y2,x3,y3,y,end1,end2);
		colCount = floor(end2) - ceil(end1) + 1;
	}
};

/*
    Given some list of groups let pred be the
    number of elements in each group, and offset
    be the starting position of each group in a
    list of all elements in a supergroup containing
    all groups. expand_int generates a list of all elements
    of the supergroup where the value at an elements indice
    is the index of the group it belongs to.
*/
/*
void expand_int
	(thrust::device_vector<int>::iterator map,
	 thrust::device_vector<int>::iterator pred,
	 thrust::device_vector<int>::iterator start,
	 thrust::device_vector<int>::iterator end,
	 int num)
{
	thrust::scatter_if
		(thrust::counting_iterator<int>(0),
		 thrust::counting_iterator<int>(num),
		 map,
		 pred,
		 start);

	thrust::inclusive_scan
		(start,
		 end,
		 start,
		 thrust::maximum<int>());
}	
*/

/*
   Take a list of values and a list of counts,
   and duplicate each value a number of times
   equal to the count at its index
*/
template<typename T, typename CountT>
void vduplicate(const viskores::cont::ArrayHandle<T> &values,
		 const viskores::cont::ArrayHandle<CountT> &count,
		 viskores::cont::ArrayHandle<T> &output)
{
	viskores::cont::Invoker invoke;
	viskores::worklet::ScatterCounting scatter(count);
	ExpandWorklet expand_worklet;
	invoke(
		expand_worklet,
		scatter,
		values,
		count,
		output
	);

}

/*
    Given some list of groups let counts be the
    number of elements in each group, and assume
    that the order in which groups appear in counts
    is the same as the order in which they appear in
    a supergroup containing all groups. vexpand generates
    a list of all elements of the supergroup where
    the value at an element's indice is the index 
    of the group it belongs to.
*/
template<typename T, typename CountT>
void vexpand(viskores::cont::ArrayHandle<CountT> &counts,
		 viskores::cont::ArrayHandle<T> &output)
{
	viskores::Id length = counts.GetNumberOfValues();
	viskores::cont::ArrayHandle<T> sequence;
	viskores::cont::ArrayCopy
		(viskores::cont::make_ArrayHandleCounting<T>(0, 1, length),
		 sequence);
	vduplicate<T, CountT>(
		sequence,
		counts,
		output
	);
}

/*
   Let map be a list associating elements to their
   groups, and src be a list of group offsets.
   index_int generates a list where the value at an
   element's index is its index within its group.
*/
/*
void index_int
	(thrust::device_vector<int>::iterator map,
	 thrust::device_vector<int>::iterator src,
	 thrust::device_vector<int>::iterator out,
	 int num)
{
	thrust::transform
		(thrust::counting_iterator<int>(0),
		 thrust::counting_iterator<int>(num),
		 thrust::make_permutation_iterator(src, map),
		 out,
		 thrust::minus<int>());
}
*/

/*
   Let map be a list associating elements to their
   groups, and src be a list of group offsets.
   vindex generates a list where the value at an
   element's index is its index within its group.
*/
template<typename IndexType, typename ValueType>
void vindex
	(viskores::cont::ArrayHandle<IndexType> &map,
	 viskores::cont::ArrayHandle<ValueType> &src,
	 viskores::cont::ArrayHandle<ValueType> &out)
{
	viskores::Id length = map.GetNumberOfValues();
	viskores::cont::Algorithm::Transform
		(viskores::cont::make_ArrayHandleCounting<ValueType>(0, 1, length),
		 viskores::cont::make_ArrayHandlePermutation(map, src),
		 out, std::minus<ValueType>());
}

/*
void print_int_vec(thrust::device_vector<int>::iterator start,
		   thrust::device_vector<int>::iterator end)
{
	for(; start < end; start++)
		std::cout << *start << " ";
	std::cout << std::endl;
}
*/

/*
void print_pair_vec(thrust::device_vector<thrust::pair<int,int>>::iterator start,
		    thrust::device_vector<thrust::pair<int,int>>::iterator end)
{
	for(; start < end; start++)
	{
		thrust::pair<int,int> temp = *start;
		std::cout << temp.first << "," << temp.second << "\t";
	}
	std::cout << std::endl;
}
*/

/*
void print_float_vec(thrust::device_vector<float>::iterator start,
		     thrust::device_vector<float>::iterator end)
{
	for(; start < end; start++)
		std::cout << *start << " ";
	std::cout << std::endl;
}
*/

template<typename T>
void print_ArrayHandle(const viskores::cont::ArrayHandle<T> &arr)
{
	auto arr_Reader = arr.ReadPortal();
	for (viskores::Id i = 0; i < arr_Reader.GetNumberOfValues(); i++)
	{
		std::cout << arr_Reader.Get(i) << "\t";
	}
	std::cout << std::endl;
}

template<typename VecType>
void print_VecArray(const viskores::cont::ArrayHandle<VecType> &arr, const viskores::Id len)
{
	auto arr_Reader = arr.ReadPortal();
	for (viskores::Id i = 0; i < arr_Reader.GetNumberOfValues(); i++)
	{
		VecType vec = arr_Reader.Get(i);
		std::cout << "( ";
		for (viskores::Id j = 0; j < len; j++)
			std::cout << vec[j] << " ";
		std::cout << ")\t";

	}
	std::cout << std::endl;
}

/*
struct key_equality
{
	__host__ __device__	
	bool operator()
		(thrust::pair<thrust::pair<int,int>, int> p1, thrust::pair<thrust::pair<int,int>, int> p2)
	{
		return thrust::get<0>(thrust::get<0>(p1)) == thrust::get<0>(thrust::get<0>(p2)) &&
		       thrust::get<1>(thrust::get<0>(p1)) == thrust::get<1>(thrust::get<0>(p2));
	}
};
*/

/*
struct findPositions
{
	thrust::device_vector<thrust::pair<int,int>>::iterator start;
	thrust::device_vector<thrust::pair<int,int>>::iterator stop;
	findPositions
		(thrust::device_vector<thrust::pair<int,int>>::iterator _start, thrust::device_vector<thrust::pair<int,int>>::iterator _stop)
		: start(_start), stop(_stop) {}

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		thrust::pair<int,int> pos = thrust::get<0>(t);
		thrust::get<1>(t) = (int)(thrust::find(start, stop, pos) - start);
	}
};
*/

struct ToRowMajor : viskores::worklet::WorkletMapField
{
	int w;
	ToRowMajor(int _w) : w(_w) {}

	using ControlSignature = void(FieldIn coordinates, FieldOut indices);
	using ExecutionSignature = _2(_1);

	VISKORES_EXEC
	int operator()(const viskores::Vec2i &pos) const
	{
		return pos[0] + pos[1] * w;
	}
};

void RasterizeTriangles(viskores::cont::ArrayHandle<viskores::Vec3f> &p1,
		viskores::cont::ArrayHandle<viskores::Vec3f> &p2,
		viskores::cont::ArrayHandle<viskores::Vec3f> &p3,
		viskores::cont::ArrayHandle<viskores::Vec3ui_8> &color,
		int numTri, int width, int height, Image &final_image)
{
	//Set up timing systems
	std::vector<std::chrono::time_point<std::chrono::high_resolution_clock>> timer;
	//time: function start
	timer.push_back(std::chrono::high_resolution_clock::now());	
	
	//Define a Viskores Invoker
	viskores::cont::Invoker invoke;

/*
   RASTERIZE
*/

#if DEBUG > 0
	std::cout << "Count fragments" << std::endl;
#endif
#if DEBUG > 1 
	std::cout << numTri << " Triangles" << std::endl;
#endif	
	viskores::cont::ArrayHandle<int> frags;

	FragCount fragCount;
	invoke(fragCount, p1, p2, p3, frags);

#if DEBUG > 1 
	std::cout << "# frags by triange: " << std::endl;
	print_ArrayHandle(frags);
#endif
	viskores::cont::ArrayHandle<viskores::Id> write_index;
	viskores::cont::Algorithm::ScanExclusive(viskores::cont::make_ArrayHandleCast<viskores::Id>(frags),
			write_index);
#if DEBUG > 1
	std::cout << "write position by triange: " << std::endl;
	print_ArrayHandle(write_index);
#endif

	int fragments = write_index.ReadPortal().Get(numTri-1) + frags.ReadPortal().Get(numTri-1);
#if DEBUG > 1	
	std::cout << "Number of fragments: " << fragments << std::endl;
#endif
#if DEBUG > 0
	std::cout << "Get fragments" << std::endl;
#endif

	viskores::cont::ArrayHandle<viskores::Id> frag_tri;
	vexpand(frags, frag_tri);
#if DEBUG > 3
	std::cout << "Which triangle does each fragment belong to?" << std::endl;
	print_ArrayHandle(frag_tri);
#endif
/*
	thrust::scatter_if
		(thrust::counting_iterator<int>(0),
		 thrust::counting_iterator<int>(2),
		 write_index.begin(),
		 frags.begin(),
		 frag_pos.begin());

	thrust::inclusive_scan
		(frag_pos.begin(),
		 frag_pos.end(),
		 frag_pos.begin(),
		 thrust::maximum<int>());
	
	thrust::device_vector<int> frag_ind(fragments);

	thrust::transform
		(thrust::counting_iterator<int>(0),
		 thrust::counting_iterator<int>(fragments),
		 thrust::make_permutation_iterator(write_index.begin(), frag_pos.begin()), frag_ind.begin(),
		 thrust::minus<int>());
*/	
	//Initialize ArrayHandles
	viskores::cont::ArrayHandle<int> rows;
	RowCount rowCount;
	invoke(rowCount, p1, p2, p3, rows);
#if DEBUG > 1
	std::cout << "How many rows does each triangle have?" << std::endl;
	print_ArrayHandle(rows);
#endif
	viskores::cont::ArrayHandle<viskores::Id> row_off;
	viskores::cont::Algorithm::ScanExclusive(viskores::cont::make_ArrayHandleCast<viskores::Id>(rows), row_off);
#if DEBUG > 1
	std::cout << "What is the row offset of each triangle?" << std::endl;
	print_ArrayHandle(row_off);
#endif

	int num_rows = row_off.ReadPortal().Get(numTri-1) + rows.ReadPortal().Get(numTri-1);

	//Initialize ArrayHandles
	viskores::cont::ArrayHandle<viskores::Id> tri_ptr;
	
	vexpand(rows, tri_ptr);
#if DEBUG > 2 
	std::cout << "What triangle does each row belong to?" << std::endl;
	print_ArrayHandle(tri_ptr);
#endif
	//Initialize ArrayHandles
	viskores::cont::ArrayHandle<viskores::Id> row_ptr;

	vindex(tri_ptr, row_off, row_ptr);
#if DEBUG > 2 
	std::cout << "The index of each row." << std::endl;
	print_ArrayHandle(row_ptr);
#endif
	//Initialize ArrayHandles
	viskores::cont::ArrayHandle<int> col_count;

	ColCount colCount;
	invoke(
		colCount,
		viskores::cont::make_ArrayHandlePermutation(tri_ptr, p1),
		viskores::cont::make_ArrayHandlePermutation(tri_ptr, p2),
		viskores::cont::make_ArrayHandlePermutation(tri_ptr, p3),
		row_ptr,
		col_count
	);
#if DEBUG > 2
	std::cout << "How many columns does each row have?" << std::endl;
	print_ArrayHandle(col_count);
#endif
	//Initialize ArrayHandles
	viskores::cont::ArrayHandle<viskores::Id> col_off;

	viskores::cont::Algorithm::ScanExclusive
		(viskores::cont::make_ArrayHandleCast<viskores::Id>(col_count), col_off);
#if DEBUG > 2 
	std::cout << "Column offsets by row" << std::endl;
	print_ArrayHandle(col_off);
	std::cout << "Number of columns " <<  col_off.ReadPortal.Get(num_rows-1) + 
		col_count.ReadPortal.Get(num_rows-1) << std::endl;
#endif
	assert((fragments == (int)col_off.ReadPortal().Get(num_rows-1) + (int)col_count.ReadPortal().Get(num_rows-1)));
	//Initialize ArrayHandles
	viskores::cont::ArrayHandle<viskores::Id> frag_row;
	viskores::cont::ArrayHandle<viskores::Id> frag_col;

	//Determine fragment rows and columns
	vexpand(col_count, frag_row);
	std::cout << "Frag Rows" << std::endl;
	//print_ArrayHandle(frag_row);

	//temporary copies
	/*
	viskores::cont::ArrayHandle<viskores::Id> vtmp_frag_row;
	viskores::cont::ArrayHandle<viskores::Id> vtmp_frag_col;
	vtmp_frag_row.DeepCopyFrom(frag_row);
	vtmp_frag_col.DeepCopyFrom(frag_col);
	*/

	vindex(frag_row, col_off, frag_col);

	viskores::cont::Algorithm::Transform
		(frag_row,
		 viskores::cont::make_ArrayHandlePermutation(frag_tri, row_off),
		 frag_row,
		 std::minus<viskores::Id>());
	//std::cout << "Size of frag_row, frag_col: " <<
	//	frag_row.GetNumberOfValues() << ", " <<
	//	frag_col.GetNumberOfValues() << std::endl;
	//std::cout << "Frag Col" << std::endl;
	//print_ArrayHandle(frag_col);
		 
#if DEBUG > 3 
	std::cout << "Frag positions by row and column in every triangle." << std::endl;
	print_ArrayHandle(frag_row);
	print_ArrayHandle(frag_col);
#endif
	//Initialize ArrayHandles
	viskores::cont::ArrayHandle<viskores::Vec2i> pos;
	viskores::cont::ArrayHandle<float> depth;

	/*thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				thrust::make_permutation_iterator(p1.begin(), frag_tri.begin()),
				thrust::make_permutation_iterator(p2.begin(), frag_tri.begin()),
				thrust::make_permutation_iterator(p3.begin(), frag_tri.begin()),
				frag_row.begin(), frag_col.begin(), pos.begin(), depth.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				thrust::make_permutation_iterator(p1.begin(), frag_tri.end()),
				thrust::make_permutation_iterator(p2.begin(), frag_tri.end()),
				thrust::make_permutation_iterator(p3.begin(), frag_tri.end()),
				frag_row.end(), frag_col.end(), pos.end(), depth.end())),
		rasterize());*/

	//Rasterize
	Rasterize rasterize;	
	invoke(rasterize,
		viskores::cont::make_ArrayHandlePermutation(frag_tri, p1),
		viskores::cont::make_ArrayHandlePermutation(frag_tri, p2),
		viskores::cont::make_ArrayHandlePermutation(frag_tri, p3),
		frag_row, frag_col, pos, depth);

	/*auto tmp_pos_Reader = pos.ReadPortal();
	for(viskores::Id i = 0; i < tmp_pos_Reader.GetNumberOfValues(); i++)
	{
		std::cout << "(" << thrust::get<0>(tmp_pos_Reader.Get(i)) << ", " << 
			thrust::get<1>(tmp_pos_Reader.Get(i)) << ")\t";
	}
	std::cout << std::endl;
	auto tmp_dep_Reader = depth.ReadPortal();
	for(viskores::Id i = 0; i < tmp_dep_Reader.GetNumberOfValues(); i++)
	{
		std::cout << tmp_dep_Reader.Get(i) << "\t";
	}
	std::cout << std::endl;*/

#if DEBUG > 3
	std::cout << "Position and depth of fragments" << std::endl;
	print_VecArray(pos, 2);
	print_ArrayHandle(depth);
#endif
	//Gather the color of each fragment
	viskores::cont::ArrayHandlePermutation<viskores::cont::ArrayHandle<viskores::Id>, viskores::cont::ArrayHandle<viskores::Vec3ui_8>> frag_colors(frag_tri, color);

	//time: rasterized triangles. acquired all fragments
	timer.push_back(std::chrono::high_resolution_clock::now());

/*
   SORT
*/

#if DEBUG > 0	
	std::cout << "find fragments to write" << std::endl;

	std::cout << "\tcopy position" << std::endl;
#endif

	//Allocate ArrayHandles for Sorting
	viskores::cont::ArrayHandle<viskores::Vec2i> cpos;
	cpos.DeepCopyFrom(pos);	
	viskores::cont::ArrayHandleCounting<viskores::Id> tmp_inds(0, 1, fragments);
	viskores::cont::ArrayHandle<viskores::Id> sorted_inds;
	viskores::cont::Algorithm::Copy(tmp_inds, sorted_inds);

#if DEBUG > 0
	std::cout << "\tsort fragments" << std::endl;
#endif
	viskores::cont::Algorithm::SortByKey(cpos, sorted_inds);
	viskores::cont::ArrayHandlePermutation<viskores::cont::ArrayHandle<viskores::Id>, viskores::cont::ArrayHandlePermutation<viskores::cont::ArrayHandle<viskores::Id>, viskores::cont::ArrayHandle<viskores::Vec3ui_8>>> cfrag_colors(sorted_inds, frag_colors);
	viskores::cont::ArrayHandlePermutation<viskores::cont::ArrayHandle<viskores::Id>, viskores::cont::ArrayHandle<float>> cdepth(sorted_inds, depth);

#if DEBUG > 3
	std::cout << "Sorted" << std::endl;
	print_VecArray(cpos, 2);
	print_ArrayHandle(sorted_inds);
	print_ArrayHandle(cdepth);
#endif
	//time: sorted fragments
	timer.push_back(std::chrono::high_resolution_clock::now());

/*
   SELECT
*/

#if DEBUG > 0
	std::cout << "\tget fragments at lowest depth" << std::endl;
#endif
/*
	//count the number of unique positions
	int unique_positions;
	{
		viskores::cont::ArrayHandle<viskores::Vec2i> tmp_pos;
		viskores::cont::Algorithm::Copy(cpos, tmp_pos);
		viskores::cont::Algorithm::Unique(tmp_pos);
		unique_positions = tmp_pos.GetNumberOfValues();
	}
#if DEBUG > 1
	std::cout << "\tunique positions = " << unique_positions << std::endl;
#endif
*/
	viskores::cont::ArrayHandle<viskores::Vec2i> true_fragments;
	viskores::cont::ArrayHandle<float> min_depth;
	viskores::cont::ArrayHandle<int> pos_count;
	viskores::cont::Algorithm::ReduceByKey(cpos, cdepth, true_fragments, min_depth, my_maximum<float>());
	viskores::cont::Algorithm::ReduceByKey(cpos, viskores::cont::make_ArrayHandleConstant<int>(1, fragments),
		       true_fragments, pos_count, std::plus<int>());	

#if DEBUG > 3
	std::cout << "Number of duplicates at each unique position" << std::endl;
	print_ArrayHandle(pos_count);
#endif
#if DEBUG > 0
	std::cout << "\tGet the minimum depth of each unique position" << std::endl;
#endif
	/* Thrust Implementation

	thrust::device_vector<int> pos_start_ind(unique_positions);
	thrust::exclusive_scan(pos_count.begin(), pos_count.end(), pos_start_ind.begin());
#if DEBUG > 3
	std::cout << "Offset by unique position" << std::endl;
	print_int_vec(pos_start_ind.begin(), pos_start_ind.end());
#endif
	thrust::device_vector<int> depth_map(fragments);
	expand_int(pos_start_ind.begin(), pos_count.begin(), depth_map.begin(), depth_map.end(), unique_positions);
#if DEBUG > 3
	std::cout << "Min depth gather position by fragment" << std::endl;
	print_int_vec(depth_map.begin(), depth_map.end());
#endif
	thrust::device_vector<float> exp_min_depth(fragments);
	thrust::gather(depth_map.begin(), depth_map.end(), min_depth.begin(), exp_min_depth.begin());

	*/

	/* Viskores Implementation */

	viskores::cont::ArrayHandle<float> exp_min_depth;
	vduplicate(min_depth, pos_count, exp_min_depth);

#if DEBUG > 3
	std::cout << "Min depth by fragment" << std::endl;
	print_ArrayHandle(exp_min_depth);
#endif
/*
	//std::cout << "Min depth" << std::endl;
	//print_pair_vec(true_fragments.begin(), true_fragments.end());
	//print_float_vec(min_depth.begin(), min_depth.end());

	//thrust::device_vector<thrust::pair<int,int>>::iterator true_end = thrust::unique(true_fragments.begin(), true_fragments.end()) - 1;
	//print_pair_vec(true_fragments.begin(), true_fragments.end());

	std::cout << "\tfor each position, get the shallowest depth of a fragment at that position" << std::endl;
	thrust::device_vector<int> find_real(fragments);
	std::cout <<"\t\tfind each fragment position in list of lowest fragment positions" << std::endl;
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(pos.begin(), find_real.begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(pos.end(), find_real.end())),
			 findPositions(true_fragments.begin(), true_fragments.end()));
	//print_int_vec(find_real.begin(), find_real.end());
	std::cout << "\t\tgather the shallowest depth for each fragment position" << std::endl;
	thrust::device_vector<float> min_depth_by_fragment(fragments);
	thrust::gather(find_real.begin(), find_real.end(), min_depth.begin(), min_depth_by_fragment.begin());
*/
#if DEBUG > 3
	std::cout << "Min Depth at fragment position vs fragment depth" << std::endl;
	print_ArrayHandle(exp_min_depth);
	print_ArrayHandle(cdepth);
#endif
#if DEBUG > 0
	std::cout << "\tchoose fragments to write" << std::endl;
#endif
	viskores::cont::ArrayHandle<bool> write_frag;
	viskores::cont::Algorithm::Transform(exp_min_depth, cdepth, write_frag, std::equal_to<float>());

	/*
	//Convert ArrayHandles to Thrust vectors

	//Create portals for reading
	auto pos_Reader = cpos.ReadPortal();
	auto depth_Reader = cdepth.ReadPortal();
	auto color_Reader = cfrag_colors.ReadPortal();
	auto inds_Reader = sorted_inds.ReadPortal();

	auto true_frag_reader = true_fragments.ReadPortal();
	auto min_depth_reader = min_depth.ReadPortal();
	auto pos_count_reader = pos_count.ReadPortal();
	auto exp_min_depth_reader = exp_min_depth.ReadPortal();
	auto write_frag_reader = write_frag.ReadPortal();

	//Create Thrust vectors
	thrust::device_vector<thrust::pair<int,int>> cpos(
		viskores::cont::ArrayPortalToIteratorBegin(pos_Reader),
		viskores::cont::ArrayPortalToIteratorEnd(pos_Reader)
	);
	thrust::device_vector<float> cdepth(
		viskores::cont::ArrayPortalToIteratorBegin(depth_Reader),
		viskores::cont::ArrayPortalToIteratorEnd(depth_Reader)
	);
	thrust::device_vector<thrust::tuple<char,char,char>> cfrag_colors(
		viskores::cont::ArrayPortalToIteratorBegin(color_Reader),
		viskores::cont::ArrayPortalToIteratorEnd(color_Reader)
	);
	thrust::device_vector<int> sorted_inds(
		viskores::cont::ArrayPortalToIteratorBegin(inds_Reader),
		viskores::cont::ArrayPortalToIteratorEnd(inds_Reader)
	);

	thrust::device_vector<thrust::pair<int,int>> true_fragments(
		viskores::cont::ArrayPortalToIteratorBegin(true_frag_reader),
		viskores::cont::ArrayPortalToIteratorEnd(true_frag_reader)
	);
	thrust::device_vector<float> min_depth(
		viskores::cont::ArrayPortalToIteratorBegin(min_depth_reader),
		viskores::cont::ArrayPortalToIteratorEnd(min_depth_reader)
	);
	thrust::device_vector<int> pos_count(
		viskores::cont::ArrayPortalToIteratorBegin(pos_count_reader),
		viskores::cont::ArrayPortalToIteratorEnd(pos_count_reader)
	);
	thrust::device_vector<float> exp_min_depth(
		viskores::cont::ArrayPortalToIteratorBegin(exp_min_depth_reader),
		viskores::cont::ArrayPortalToIteratorEnd(exp_min_depth_reader)
	);
	thrust::device_vector<bool> write_frag(
		viskores::cont::ArrayPortalToIteratorBegin(write_frag_reader),
		viskores::cont::ArrayPortalToIteratorEnd(write_frag_reader)
	);
	*/

#if DEBUG > 3
	std::cout << "Write fragment?" << std::endl;
	print_ArrayHandle(write_frag);
#endif
	//time: got visible fragments
	timer.push_back(std::chrono::high_resolution_clock::now());

/*
   WRITE
*/

#if DEBUG > 0
	std::cout << "write fragments" << std::endl;
#endif

	viskores::cont::ArrayHandle<viskores::Id> rowMajorPos;
	ToRowMajor to_row_major(width);
	invoke(to_row_major, cpos, rowMajorPos);

#if DEBUG > 3
	std::cout << "Row major position by fragment" << std::endl;
	print_ArrayHandle(rowMajorPos);
#endif

	//viskores::cont::ArrayHandle<viskores:Vec3ui_8> vbg;
	//vbg.AllocateAndFill(width * height, thrust::make_tuple<char,char,char>(127,127,127));
	viskores::cont::ArrayHandle<viskores::Vec3ui_8> img;
	img.AllocateAndFill(width * height, viskores::make_Vec<viskores::UInt8>(127,127,127));
	/*
	std::cout << cfrag_colors.GetNumberOfValues() << std::endl;
	std::cout << rowMajorPos.GetNumberOfValues() << std::endl;
	std::cout << write_frag.GetNumberOfValues() << std::endl;
	std::cout << img.GetNumberOfValues() << std::endl;
	*/
	//auto max_pos = viskores::cont::Algorithm::Reduce(rowMajorPos, (viskores::Id) 0,
	//	       [](const auto& a, const auto& b){return std::max(a,b);});	
	//std::cout << max_pos << std::endl;
	FillImage<viskores::cont::StorageTagBasic> fill_image;
	invoke(
		fill_image,
		cfrag_colors,
		rowMajorPos,
		write_frag,
		img
	);

	auto img_Reader = img.ReadPortal();
	int count = 0;
	for(viskores::Id i = 0; i < img_Reader.GetNumberOfValues(); i++)
	{
		viskores::Vec3ui_8 t = img_Reader.Get(i);
		final_image.data[count++] = t[0];
		final_image.data[count++] = t[1];
		final_image.data[count++] = t[2];
	}
	//time: write final image to output
	timer.push_back(std::chrono::high_resolution_clock::now());

/*
   DONE
*/

	//char *col = final_image.data;
	//for(int i = 0; i < 60; i+=3)
	//{
	//	std::cout<<(int)col[i]<<","<<(int)col[i+1]<<","<<(int)col[i+2]<<std::endl;
	//}
	auto p = timer.begin();
	for(auto i = timer.begin() + 1; i != timer.end(); i++)
	{
		auto duration = std::chrono::duration_cast<std::chrono::microseconds>(*i - *p);
		p = i;
		std::cout << "\t" << duration.count();	
	}
	std::cout << std::endl;
}
