#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/iterator/zip_iterator.h>

#include <viskores/cont/Initialize.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <cstdlib>
#include <string>

#include "rastByTri.h"
#include "imageWriter.h"
#include "readSTL.h"



#ifndef DEBUG
#define DEBUG 0
#endif

void parseTriPair(const std::string &str, float &v1, float &v2, float &v3)
{
	//std::cout << "parsing " << str << std::endl;
	/*std::stringstream ss;
	ss << str;
	//int tmp;
	
	std::cout << "parsed ";
	for(; ss >> tmp;)
	{
		std::cout  << tmp << " ";
		if(ss.peek() == ',')
			ss.ignore();
	}
	std::cout << std::endl;
	if(!(ss >> v1))
	{
		std::cerr << "Could not parse values from triangle input file to integers" << std::endl;
		exit(EXIT_FAILURE);
	}
	if(ss.peek() == ',')
		ss.ignore();
	if(!(ss >> v2))
	{
		std::cerr << "Could not parse values from triangle input file to integers" << std::endl;
		exit(EXIT_FAILURE);
	}
	if(ss.peek() == ',')
		ss.ignore();
	if(!(ss >> v3))
	{
		std::cerr << "Could not parse values from triangle input file to integers" << std::endl;
		exit(EXIT_FAILURE);
	}*/
	char *temp_line = new char [str.length() + 1];
	strcpy(temp_line, str.c_str());
	char *tok = strtok(temp_line, ",");
	//std::cout << tok << ",";
	v1 = atoi(tok);
	tok = strtok(NULL, ",");
	//std::cout << tok << ",";
	v2 = atoi(tok);
	tok = strtok(NULL, ",");
	//std::cout << tok << ",";
	v3 = atoi(tok);
}

void readTriangles(thrust::device_vector<thrust::tuple<float,float,float>> &p1,
		thrust::device_vector<thrust::tuple<float,float,float>> &p2,
		thrust::device_vector<thrust::tuple<float,float,float>> &p3,
		thrust::device_vector<thrust::tuple<char,char,char>> &color,
		int &numTri, char *filename, int &width, int &height)
{
	std::cout << "Start Read Triangles" << std::endl;
	std::ifstream fin(filename);
	std::stringstream ss;
	std::string l1, l2, l3, l4;
	getline(fin, l1);
	ss << l1;
	ss >> numTri;
	//resize the output vectors
	p1.resize(numTri);
	p2.resize(numTri);
	p3.resize(numTri);
	color.resize(numTri);
	//create reading vectors
	//p1
	thrust::host_vector<float> p11(numTri);
	thrust::host_vector<float> p12(numTri);
	thrust::host_vector<float> p13(numTri);
	//p2
	thrust::host_vector<float> p21(numTri);
	thrust::host_vector<float> p22(numTri);
	thrust::host_vector<float> p23(numTri);
	//p3
	thrust::host_vector<float> p31(numTri);
	thrust::host_vector<float> p32(numTri);
	thrust::host_vector<float> p33(numTri);
	//color
	thrust::host_vector<char> c1(numTri);
	thrust::host_vector<char> c2(numTri);
	thrust::host_vector<char> c3(numTri);
	std::cout << numTri << " Triangles" << std::endl;
	//parse file
	//read contents into linear vectors
	//variables to record minimum x and y values
	int lowx = 0;
	int lowy = 0;
	for(int i = 0; i < numTri; i++)
	{
		//if(i % 10000 == 0)
			//std::cout << "Parsing Triangle " << i << std::endl;
		if(!getline(fin, l1) || !getline(fin, l2) || !getline(fin, l3) || !getline(fin, l4))
		{
			std::cout<<"Not enough values in "<<filename<<" for "<<numTri<<" triangles!" <<std::endl;
			exit(EXIT_FAILURE);
		}
		float v1, v2, v3;
		//std::cout << "color " << l1 << std::endl;
		parseTriPair(l1, v1, v2, v3);
		//std::cout <<"parsed "<< v1 << ", " << v2 << ", " << v3 << std::endl;
		//color[i] = thrust::make_tuple<char,char,char>((char)v1,(char)v2,(char)v3);
		c1[i] = char(v1);
		c2[i] = char(v2);
		c3[i] = char(v3);
		//std::cout << "p1 " << l2 << std::endl;
		parseTriPair(l2, v1, v2, v3);
		//std::cout <<"parsed "<< v1 << ", " << v2 << ", " << v3 << std::endl;
		//p1[i] = thrust::make_tuple<float,float,float>(v1,v2,v3);
		p11[i] = v1; //get the x position 
		p12[i] = v2; //get the y position
		p13[i] = v3; //get the z position
		width = std::max(width, (int) v1 + 1); //update width
		height = std::max(height, (int) v2 + 1); //update height
		lowx = std::min(lowx, (int) v1); //update lowx
		lowy = std::min(lowy, (int) v2); //update lowy
		//std::cout << "p2 " << l3 << std::endl;
		parseTriPair(l3, v1, v2, v3);
		//std::cout <<"parsed "<< v1 << ", " << v2 << ", " << v3 << std::endl;
		//p2[i] = thrust::make_tuple<float,float,float>(v1,v2,v3);
		p21[i] = v1;
		p22[i] = v2;
		p23[i] = v3;
		width = std::max(width, (int) v1 + 1);
		height = std::max(height, (int) v2 + 1);
		lowx = std::min(lowx, (int) v1);
		lowy = std::min(lowy, (int) v2);
		//std::cout << "p3 " << l4 << std::endl;
		parseTriPair(l4, v1, v2, v3);
		//std::cout <<"parsed "<< v1 << ", " << v2 << ", " << v3 << std::endl;
		//p3[i] = thrust::make_tuple<float,float,float>(v1,v2,v3);
		p31[i] = v1;
		p32[i] = v2;
		p33[i] = v3;
		width = std::max(width, (int) v1 + 1);
		height = std::max(height, (int) v2 + 1);
		lowx = std::min(lowx, (int) v1);
		lowy = std::min(lowy, (int) v2);

	}
	fin.close();

	auto p1b = thrust::make_zip_iterator(thrust::make_tuple(p11.begin(), p12.begin(), p13.begin()));
	auto p1e = thrust::make_zip_iterator(thrust::make_tuple(p11.end(), p12.end(), p13.end()));
	thrust::copy(p1b, p1e, p1.begin());
	auto p2b = thrust::make_zip_iterator(thrust::make_tuple(p21.begin(), p22.begin(), p23.begin()));
	auto p2e = thrust::make_zip_iterator(thrust::make_tuple(p21.end(), p22.end(), p23.end()));
	thrust::copy(p2b, p2e, p2.begin());
	auto p3b = thrust::make_zip_iterator(thrust::make_tuple(p31.begin(), p32.begin(), p33.begin()));
	auto p3e = thrust::make_zip_iterator(thrust::make_tuple(p31.end(), p32.end(), p33.end()));
	thrust::copy(p3b, p3e, p3.begin());
	auto cb = thrust::make_zip_iterator(thrust::make_tuple(c1.begin(), c2.begin(), c3.begin()));
	auto ce = thrust::make_zip_iterator(thrust::make_tuple(c1.end(), c2.end(), c3.end()));
	thrust::copy(cb, ce, color.begin());
	width -= lowx;
	height -= lowy;
}
	

int main(int argc, char **argv)
{
	//initialize viskores
	viskores::cont::Initialize(argc, argv, viskores::cont::InitializeOptions::AddHelp);

	if(argc < 3)
	{
		std::cerr << "USAGE: rast <input> <output> " << std::endl;
		exit(EXIT_FAILURE);
	}

	//create width and height variables
	int WIDTH = 300;
	int HEIGHT = 300;

#if DEBUG > 0
	std::cout << "initialize triangles" << std::endl;
#endif
	thrust::device_vector<thrust::tuple<float, float, float>> p1;
	thrust::device_vector<thrust::tuple<float, float, float>> p2;
	thrust::device_vector<thrust::tuple<float, float, float>> p3;
	thrust::device_vector<thrust::tuple<char, char, char>> color;

	/*p1[0] = thrust::make_tuple(0,0,0);
	p2[0] = thrust::make_tuple(5,5,0);
	p3[0] = thrust::make_tuple(10,0,0);
	p1[1] = thrust::make_tuple(10,0,-0.5);
	p2[1] = thrust::make_tuple(15,15,-0.5);
	p3[1] = thrust::make_tuple(20,0,-0.5);
	//p1[2] = thrust::make_tuple(20,0,-1);
	//p2[2] = thrust::make_tuple(25,25,-1);
	//p3[2] = thrust::make_tuple(30,0,-1);

	color[0] = thrust::make_tuple(255,0,0);
	color[1] = thrust::make_tuple(0,0,255);
	//color[2] = thrust::make_tuple(0,255,0);*/

#if DEBUG > 0
	std::cout << "Start Main" << std::endl;
#endif

	int numTri;
	char *fileType = argv[1];
	for(; (*fileType) != '.'; fileType++);
	fileType++;
#if DEBUG > 1
	std::cout << fileType << std::endl;
#endif
	if(strcmp(fileType, "tri") == 0)
		readTriangles(p1, p2, p3, color, numTri, argv[1], WIDTH, HEIGHT);
	else if(strcmp(fileType, "stl") == 0)
		numTri = readTriFromBinarySTL(p1, p2, p3, color, argv[1], WIDTH, HEIGHT);
	else
		return -1;

#if DEBUG > 0
	std::cout << "Finished Read Triangles" << std::endl;
#if DEBUG > 1	
	std::cout << "width: " << WIDTH << " height: " << HEIGHT << std::endl;

	std::cout << "Triangles: " << numTri << std::endl;
#endif
#endif
	Image final_image;
	initImage(&final_image, WIDTH, HEIGHT);
#if DEBUG > 0
	std::cout << "Finished Initialize Image" << std::endl;
#endif
	viskores::cont::ArrayHandle<thrust::tuple<float,float,float>> vp1 = 
		viskores::cont::make_ArrayHandle(thrust::raw_pointer_cast(p1.data()), p1.size(), viskores::CopyFlag::On);
	viskores::cont::ArrayHandle<thrust::tuple<float,float,float>> vp2 = 
		viskores::cont::make_ArrayHandle(thrust::raw_pointer_cast(p2.data()), p2.size(), viskores::CopyFlag::On);
	viskores::cont::ArrayHandle<thrust::tuple<float,float,float>> vp3 = 
		viskores::cont::make_ArrayHandle(thrust::raw_pointer_cast(p3.data()), p3.size(), viskores::CopyFlag::On);
	viskores::cont::ArrayHandle<thrust::tuple<char,char,char>> vcolor = 
		viskores::cont::make_ArrayHandle(thrust::raw_pointer_cast(color.data()), color.size(), viskores::CopyFlag::On);

	RasterizeTriangles(vp1, vp2, vp3, vcolor, numTri, WIDTH, HEIGHT, final_image);

#if DEBUG > 0
	std::cout << "Finished Rasterize Triangles" << std::endl;
#endif

	writeImage(&final_image, argv[2]);
	//char *col = final_image.data;
	//for(int i = 0; i < 60; i+=3)
	//{
	//	std::cout<<(int)col[i]<<","<<(int)col[i+1]<<","<<(int)col[i+2]<<std::endl;
	//}

#if DEBUG > 0
	std::cout << "Finished Write Image" << std::endl;
#endif
		
	freeImage(&final_image);

#if DEBUG > 0
	std::cout << "Program End" << std::endl;
#endif
}
