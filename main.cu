#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/iterator/zip_iterator.h>

#include <viskores/cont/Initialize.h>
#include <viskores/cont/ArrayHandle.h>
#include <viskores/Types.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <cstdlib>
#include <string>

#include "rastByTri.h"
#include "imageWriter.h"
#include "readSTL.h"

#ifndef DEBUG
#define DEBUG 0
#endif

void parseTriPair(const std::string &str, float &v1, float &v2, float &v3)
{
	//std::cout << "parsing " << str << std::endl;
	/*std::stringstream ss;
	ss << str;
	//int tmp;
	
	std::cout << "parsed ";
	for(; ss >> tmp;)
	{
		std::cout  << tmp << " ";
		if(ss.peek() == ',')
			ss.ignore();
	}
	std::cout << std::endl;
	if(!(ss >> v1))
	{
		std::cerr << "Could not parse values from triangle input file to integers" << std::endl;
		exit(EXIT_FAILURE);
	}
	if(ss.peek() == ',')
		ss.ignore();
	if(!(ss >> v2))
	{
		std::cerr << "Could not parse values from triangle input file to integers" << std::endl;
		exit(EXIT_FAILURE);
	}
	if(ss.peek() == ',')
		ss.ignore();
	if(!(ss >> v3))
	{
		std::cerr << "Could not parse values from triangle input file to integers" << std::endl;
		exit(EXIT_FAILURE);
	}*/
	char *temp_line = new char [str.length() + 1];
	strcpy(temp_line, str.c_str());
	char *tok = strtok(temp_line, ",");
	//std::cout << tok << ",";
	v1 = atoi(tok);
	tok = strtok(NULL, ",");
	//std::cout << tok << ",";
	v2 = atoi(tok);
	tok = strtok(NULL, ",");
	//std::cout << tok << ",";
	v3 = atoi(tok);
}

void readTriangles(viskores::cont::ArrayHandle<viskores::Vec3f> &p1,
		viskores::cont::ArrayHandle<viskores::Vec3f> &p2,
		viskores::cont::ArrayHandle<viskores::Vec3f> &p3,
		viskores::cont::ArrayHandle<viskores::Vec3ui_8> &color,
		int &numTri, char *filename, int &width, int &height)
{
	std::cout << "Start Read Triangles" << std::endl;
	std::ifstream fin(filename);
	std::stringstream ss;
	std::string l1, l2, l3, l4;
	getline(fin, l1);
	ss << l1;
	ss >> numTri;
	//resize the output vectors
	p1.Allocate(numTri);
	p2.Allocate(numTri);
	p3.Allocate(numTri);
	color.Allocate(numTri);
	//create reading vectors
	//p1
	auto p1_Writer = p1.WritePortal();
	//p2
	auto p2_Writer = p2.WritePortal();
	//p3
	auto p3_Writer = p3.WritePortal();
	//color
	auto color_Writer = color.WritePortal();
	std::cout << numTri << " Triangles" << std::endl;
	//parse file
	//read contents into linear vectors
	//variables to record minimum x and y values
	int lowx = 0;
	int lowy = 0;
	for(int i = 0; i < numTri; i++)
	{
		//if(i % 10000 == 0)
			//std::cout << "Parsing Triangle " << i << std::endl;
		if(!getline(fin, l1) || !getline(fin, l2) || !getline(fin, l3) || !getline(fin, l4))
		{
			std::cout<<"Not enough values in "<<filename<<" for "<<numTri<<" triangles!" <<std::endl;
			exit(EXIT_FAILURE);
		}
		float v1, v2, v3;
		//std::cout << "color " << l1 << std::endl;
		parseTriPair(l1, v1, v2, v3);
		//std::cout <<"parsed "<< v1 << ", " << v2 << ", " << v3 << std::endl;
		//color[i] = thrust::make_tuple<char,char,char>((char)v1,(char)v2,(char)v3);
		color_Writer.Set(i, viskores::make_Vec(char(v1), char(v2), char(v3)));
		//std::cout << "p1 " << l2 << std::endl;
		parseTriPair(l2, v1, v2, v3);
		//std::cout <<"parsed "<< v1 << ", " << v2 << ", " << v3 << std::endl;
		//p1[i] = thrust::make_tuple<float,float,float>(v1,v2,v3);
		p1_Writer.Set(i, viskores::make_Vec(v1, v2, v3));
		width = std::max(width, (int) v1 + 1); //update width
		height = std::max(height, (int) v2 + 1); //update height
		lowx = std::min(lowx, (int) v1); //update lowx
		lowy = std::min(lowy, (int) v2); //update lowy
		//std::cout << "p2 " << l3 << std::endl;
		parseTriPair(l3, v1, v2, v3);
		//std::cout <<"parsed "<< v1 << ", " << v2 << ", " << v3 << std::endl;
		//p2[i] = thrust::make_tuple<float,float,float>(v1,v2,v3);
		p2_Writer.Set(i, viskores::make_Vec(v1, v2, v3));
		width = std::max(width, (int) v1 + 1); //update width
		width = std::max(width, (int) v1 + 1);
		height = std::max(height, (int) v2 + 1);
		lowx = std::min(lowx, (int) v1);
		lowy = std::min(lowy, (int) v2);
		//std::cout << "p3 " << l4 << std::endl;
		parseTriPair(l4, v1, v2, v3);
		//std::cout <<"parsed "<< v1 << ", " << v2 << ", " << v3 << std::endl;
		//p3[i] = thrust::make_tuple<float,float,float>(v1,v2,v3);
		p3_Writer.Set(i, viskores::make_Vec(v1, v2, v3));
		width = std::max(width, (int) v1 + 1);
		height = std::max(height, (int) v2 + 1);
		lowx = std::min(lowx, (int) v1);
		lowy = std::min(lowy, (int) v2);

	}
	fin.close();

	/*
	auto p1b = thrust::make_zip_iterator(thrust::make_tuple(p11.begin(), p12.begin(), p13.begin()));
	auto p1e = thrust::make_zip_iterator(thrust::make_tuple(p11.end(), p12.end(), p13.end()));
	thrust::copy(p1b, p1e, p1.begin());
	auto p2b = thrust::make_zip_iterator(thrust::make_tuple(p21.begin(), p22.begin(), p23.begin()));
	auto p2e = thrust::make_zip_iterator(thrust::make_tuple(p21.end(), p22.end(), p23.end()));
	thrust::copy(p2b, p2e, p2.begin());
	auto p3b = thrust::make_zip_iterator(thrust::make_tuple(p31.begin(), p32.begin(), p33.begin()));
	auto p3e = thrust::make_zip_iterator(thrust::make_tuple(p31.end(), p32.end(), p33.end()));
	thrust::copy(p3b, p3e, p3.begin());
	auto cb = thrust::make_zip_iterator(thrust::make_tuple(c1.begin(), c2.begin(), c3.begin()));
	auto ce = thrust::make_zip_iterator(thrust::make_tuple(c1.end(), c2.end(), c3.end()));
	thrust::copy(cb, ce, color.begin());
	*/
	width -= lowx;
	height -= lowy;
}

void convertVecToTupleFloat(const viskores::cont::ArrayHandle<viskores::Vec3f> &in,
		viskores::cont::ArrayHandle<thrust::tuple<float, float, float>> &out)
{
	viskores::Id len = in.GetNumberOfValues();
	auto in_read = in.ReadPortal();

	out.Allocate(len);
	auto out_write = out.WritePortal();

	for(viskores::Id i = 0; i < len; i++)
	{
		viskores::Vec3f vec = in_read.Get(i);
		out_write.Set(i, thrust::make_tuple(vec[0], vec[1], vec[2]));
	}
}
	
void convertVecToTupleChar(const viskores::cont::ArrayHandle<viskores::Vec3ui_8> &in,
		viskores::cont::ArrayHandle<thrust::tuple<char, char, char>> &out)
{
	viskores::Id len = in.GetNumberOfValues();
	auto in_read = in.ReadPortal();

	out.Allocate(len);
	auto out_write = out.WritePortal();

	for(viskores::Id i = 0; i < len; i++)
	{
		viskores::Vec3f vec = in_read.Get(i);
		out_write.Set(i, thrust::make_tuple((char)vec[0], (char)vec[1], (char)vec[2]));
	}
}

int main(int argc, char **argv)
{
	//initialize viskores
	viskores::cont::Initialize(argc, argv, viskores::cont::InitializeOptions::AddHelp);

	if(argc < 3)
	{
		std::cerr << "USAGE: rast <input> <output> " << std::endl;
		exit(EXIT_FAILURE);
	}

	//create width and height variables
	int WIDTH = 300;
	int HEIGHT = 300;

#if DEBUG > 0
	std::cout << "initialize triangles" << std::endl;
#endif
	viskores::cont::ArrayHandle<viskores::Vec3f> p1;
	viskores::cont::ArrayHandle<viskores::Vec3f> p2;
	viskores::cont::ArrayHandle<viskores::Vec3f> p3;
	viskores::cont::ArrayHandle<viskores::Vec3ui_8> color;

	/*p1[0] = thrust::make_tuple(0,0,0);
	p2[0] = thrust::make_tuple(5,5,0);
	p3[0] = thrust::make_tuple(10,0,0);
	p1[1] = thrust::make_tuple(10,0,-0.5);
	p2[1] = thrust::make_tuple(15,15,-0.5);
	p3[1] = thrust::make_tuple(20,0,-0.5);
	//p1[2] = thrust::make_tuple(20,0,-1);
	//p2[2] = thrust::make_tuple(25,25,-1);
	//p3[2] = thrust::make_tuple(30,0,-1);

	color[0] = thrust::make_tuple(255,0,0);
	color[1] = thrust::make_tuple(0,0,255);
	//color[2] = thrust::make_tuple(0,255,0);*/

#if DEBUG > 0
	std::cout << "Start Main" << std::endl;
#endif

	int numTri;
	char *fileType = argv[1];
	for(; (*fileType) != '.'; fileType++);
	fileType++;
#if DEBUG > 1
	std::cout << fileType << std::endl;
#endif
	viskores::cont::ArrayHandle<thrust::tuple<float,float,float>> vp1; 
	viskores::cont::ArrayHandle<thrust::tuple<float,float,float>> vp2;
	viskores::cont::ArrayHandle<thrust::tuple<float,float,float>> vp3;
	viskores::cont::ArrayHandle<thrust::tuple<char,char,char>> vcolor;

	if(strcmp(fileType, "tri") == 0)
	{
		readTriangles(p1, p2, p3, color, numTri, argv[1], WIDTH, HEIGHT);
		convertVecToTupleFloat(p1, vp1);
		convertVecToTupleFloat(p2, vp2);
		convertVecToTupleFloat(p2, vp2);
		convertVecToTupleChar(color, vcolor);
	}
	else if(strcmp(fileType, "stl") == 0)
		numTri = readTriFromBinarySTL(vp1, vp2, vp3, vcolor, argv[1], WIDTH, HEIGHT);
	else
		return -1;

	/*
	std::cout << "P1 LENGTH: " << vp1.GetNumberOfValues() << std::endl;
	std::cout << "P2 LENGTH: " << vp2.GetNumberOfValues() << std::endl;
	std::cout << "P3 LENGTH: " << vp3.GetNumberOfValues() << std::endl;
	std::cout << "COLOR LENGTH: " << vcolor.GetNumberOfValues() << std::endl;
	*/

#if DEBUG > 0
	std::cout << "Finished Read Triangles" << std::endl;
#if DEBUG > 1	
	std::cout << "width: " << WIDTH << " height: " << HEIGHT << std::endl;

	std::cout << "Triangles: " << numTri << std::endl;
#endif
#endif
	Image final_image;
	initImage(&final_image, WIDTH, HEIGHT);
#if DEBUG > 0
	std::cout << "Finished Initialize Image" << std::endl;
#endif
	RasterizeTriangles(vp1, vp2, vp3, vcolor, numTri, WIDTH, HEIGHT, final_image);

#if DEBUG > 0
	std::cout << "Finished Rasterize Triangles" << std::endl;
#endif

	writeImage(&final_image, argv[2]);
	//char *col = final_image.data;
	//for(int i = 0; i < 60; i+=3)
	//{
	//	std::cout<<(int)col[i]<<","<<(int)col[i+1]<<","<<(int)col[i+2]<<std::endl;
	//}

#if DEBUG > 0
	std::cout << "Finished Write Image" << std::endl;
#endif
		
	freeImage(&final_image);

#if DEBUG > 0
	std::cout << "Program End" << std::endl;
#endif
}
